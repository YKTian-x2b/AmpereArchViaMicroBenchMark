#include "hip/hip_runtime.h"
#include "cuptr.hpp"
#include "hostptr.hpp"
#include "hip/hip_runtime_api.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

// 如果用物理地址访问L1cache，则会用到L1TLB，就会存在TLB未命中的情况。 L1TLB的覆盖范围大概是32MB。
// 开一个远大于 L1TLB 覆盖范围的数组 用足够大的步幅将要访问的元素缓存在L1cache中 但元素间隔会使 L1TLB 未命中
// 观察是否存在 L1TLB 未命中的情况，以确定L1cache indexed by 虚拟地址还是物理地址
#define GRD_SIZE 1
#define BLK_SIZE 32
// [10, 30, 50, 100, 200, 500]
#define UNROLL 100
#define WARM_UP 100
// 字节步幅为 L1 TLB entry大小 大概是2MB 
#define STRIDE (256 * 1024)

__device__ __forceinline__
void ldg_ca(void **&ldg_ptr) {
    asm volatile (
        "ld.global.ca.b64 %0, [%0];\n"
        : "+l"(ldg_ptr)
        : : "memory"
    );
}

__device__ __forceinline__
void ldg_cg(void **&ldg_ptr) {
    asm volatile (
        "ld.global.cg.b64 %0, [%0];\n"
        : "+l"(ldg_ptr)
        : : "memory"
    );
}

__global__ void L1_TLB_detect_kernel(uint32_t *startClk, uint32_t *stopClk, void **out, void **arr){
    int tid = threadIdx.x;
    void **ldg_ptr = arr +  tid;
    // populate L1 cache
    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        // 这里如果用ca结果无法解释,用cg可以看到类似TuringT4的情况
        // ldg_ca(ldg_ptr);
        ldg_cg(ldg_ptr);
    } 

    ldg_ptr = arr+tid;
    uint32_t start, stop;
    asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");

    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        // 这里如果用ca结果无法解释,用cg可以看到类似TuringT4的情况
        // ldg_ca(ldg_ptr);
        ldg_cg(ldg_ptr);
    }  
    
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid] = stop;

    if(ldg_ptr == nullptr){
        *out = ldg_ptr;
    }
}

void L1_TLB_detect_run(uint32_t ARR_SIZE_IN_K){
    uint32_t ARR_SIZE = 1024 * ARR_SIZE_IN_K;

    CuPtr<void*> arr_d(ARR_SIZE);
    HostPtr<void*> arr_h(ARR_SIZE);
    void **h_ptr = arr_h.GetPtr();
    void **d_ptr = arr_d.GetPtr();
    for(int i = 0; i < ARR_SIZE; i++){
        uint32_t offset = (i + STRIDE) % ARR_SIZE;
        h_ptr[i] = (void*)(d_ptr + offset);
    }
    uint32_t arrBytes = ARR_SIZE * sizeof(void*);
    checkCudaErrors(hipMemcpy(d_ptr, h_ptr, arrBytes, hipMemcpyHostToDevice));

    CuPtr<void*> out_d(BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);

    // warmup i-cache
    for(int i = 0; i < WARM_UP; i++){
        L1_TLB_detect_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    }
    hipDeviceSynchronize();

    // 
    L1_TLB_detect_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();

    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);

    double clockCycles_avg = 0.0;
    for(int i = 0; i < BLK_SIZE; i++){
        // std::cout << startClk_h(i) << " " << stopClk_h(i) << std::endl;
        clockCycles_avg += stopClk_h(i) - startClk_h(i);
    }
    clockCycles_avg /= (BLK_SIZE * UNROLL);
    std::cout << "ARR_SIZE_IN_MB: " << (ARR_SIZE_IN_K * 8 / 1024) << ", Global Mem access latency: " << clockCycles_avg << " cycles" << std::endl;
    
}

int main(){
    // 一个元素8B 数组字节大小上限为512MB 元素上限为64M
    // 从128K个元素(1MB) 每次递增256K(2MB) 一直到128*128K个(128MB)
    for(uint32_t arr_size = 128; arr_size < 128 * 128; arr_size += 256){
        L1_TLB_detect_run(arr_size);
    }
    // 从128*128 K个元素(128MB) 每次递增1024K(8MB) 一直到64M(512MB)个
    for(uint32_t arr_size = 128 * 128; arr_size < 64 * 1024; arr_size +=1024){
        L1_TLB_detect_run(arr_size);
    }
}


// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L1_TLB_detect L1_TLB_detect.cu

// python3 L1_TLB_detect_res_process.py
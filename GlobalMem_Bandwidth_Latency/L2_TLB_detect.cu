#include "hip/hip_runtime.h"
#include "cuptr.hpp"
#include "hostptr.hpp"
#include "hip/hip_runtime_api.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

#define GRD_SIZE 1
#define BLK_SIZE 32
#define UNROLL 100
#define WARM_UP 100
// 字节步幅为32MB 
#define STRIDE (4 * 1024 * 1024)

__device__ __forceinline__
void ldg_cg(void **&ldg_ptr) {
    asm volatile (
        "ld.global.cg.b64 %0, [%0];\n"
        : "+l"(ldg_ptr)
        : : "memory"
    );
}

__global__ void L2_TLB_detect_kernel(uint32_t *startClk, uint32_t *stopClk, void **out, void **arr){
    int tid = threadIdx.x;
    void **ldg_ptr = arr +  tid;
    // populate L2 cache
    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        ldg_cg(ldg_ptr);
    } 

    ldg_ptr = arr+tid;
    uint32_t start, stop;
    asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");

    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        ldg_cg(ldg_ptr);
    }  
    
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid] = stop;

    if(ldg_ptr == nullptr){
        *out = ldg_ptr;
    }
}

void L2_TLB_detect_run(uint32_t ARR_SIZE_IN_M){
    if(ARR_SIZE_IN_M > 384){
        // 用两次memcpy将数组传递到全局内存中
        uint32_t ARR_SIZE_one = 384 * 1024 * 1024;
        uint32_t ARR_SIZE_two = (ARR_SIZE_IN_M - 384) * 1024 * 1024;
        CuPtr<void*> arr_one_d(ARR_SIZE_one);
        CuPtr<void*> arr_two_d(ARR_SIZE_two);
        HostPtr<void*> arr_one_h(ARR_SIZE_one);
        HostPtr<void*> arr_two_h(ARR_SIZE_two);
        void **arr_one_h_ptr = arr_one_h.GetPtr();
        void **arr_two_h_ptr = arr_two_h.GetPtr();
        void **arr_one_d_ptr = arr_one_d.GetPtr();
        void **arr_two_d_ptr = arr_two_d.GetPtr();

        // // 这俩数组就是连续的
        // printf("arr_one_d_ptr: %p\n", arr_one_d_ptr);
        // printf("arr_two_d_ptr: %p\n", arr_two_d_ptr);

        // 数组1的最后STRIDE个元素指向数组2的前STRIDE个元素
        for(int i = 0; i < ARR_SIZE_one; i++){
            uint32_t offset = i + STRIDE;
            if(offset >= ARR_SIZE_one){
                arr_one_h_ptr[i] = (void*)(arr_two_d_ptr + (offset - ARR_SIZE_one));
            }
            else{
                arr_one_h_ptr[i] = (void*)(arr_one_d_ptr + offset);
            }
        }
        // 数组2的最后STRIDE个元素指向数组1的前STRIDE个元素
        for(int i = 0; i < ARR_SIZE_two; i++){
            uint32_t offset = i + STRIDE;
            if(offset >= ARR_SIZE_two){
                arr_two_h_ptr[i] = (void*)(arr_one_d_ptr + (offset - ARR_SIZE_two));
            }
            else{
                arr_two_h_ptr[i] = (void*)(arr_two_d_ptr + offset);
            }
        }
        uint32_t arrBytes_one = ARR_SIZE_one * sizeof(void*);
        uint32_t arrBytes_two = ARR_SIZE_two * sizeof(void*);
        checkCudaErrors(hipMemcpy(arr_one_d_ptr, arr_one_h_ptr, arrBytes_one, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(arr_two_d_ptr, arr_two_h_ptr, arrBytes_two, hipMemcpyHostToDevice));
        
        CuPtr<void*> out_d(BLK_SIZE);
        CuPtr<uint32_t> startClk_d(BLK_SIZE);
        CuPtr<uint32_t> stopClk_d(BLK_SIZE);

        // warmup i-cache
        for(int i = 0; i < WARM_UP; i++){
            L2_TLB_detect_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_two_d_ptr);
        }
        hipDeviceSynchronize();

        // 
        L2_TLB_detect_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_two_d_ptr);
        hipDeviceSynchronize();

        HostPtr<uint32_t> startClk_h;
        HostPtr<uint32_t> stopClk_h;
        startClk_d.ToHostPtr(startClk_h);
        stopClk_d.ToHostPtr(stopClk_h);

        double clockCycles_avg = 0.0;
        for(int i = 0; i < BLK_SIZE; i++){
            // std::cout << startClk_h(i) << " " << stopClk_h(i) << std::endl;
            clockCycles_avg += stopClk_h(i) - startClk_h(i);
        }
        clockCycles_avg /= (BLK_SIZE * UNROLL);
        std::cout << "ARR_SIZE_IN_MB: " << (ARR_SIZE_IN_M * 8) << ", Global Mem access latency: " << clockCycles_avg << " cycles" << std::endl;
    }
    else{
        uint32_t ARR_SIZE = ARR_SIZE_IN_M * 1024 * 1024;

        CuPtr<void*> arr_d(ARR_SIZE);
        HostPtr<void*> arr_h(ARR_SIZE);
        void **h_ptr = arr_h.GetPtr();
        void **d_ptr = arr_d.GetPtr();
        for(int i = 0; i < ARR_SIZE; i++){
            uint32_t offset = (i + STRIDE) % ARR_SIZE;
            h_ptr[i] = (void*)(d_ptr + offset);
        }
        uint32_t arrBytes = ARR_SIZE * sizeof(void*);
        checkCudaErrors(hipMemcpy(d_ptr, h_ptr, arrBytes, hipMemcpyHostToDevice));

        CuPtr<void*> out_d(BLK_SIZE);
        CuPtr<uint32_t> startClk_d(BLK_SIZE);
        CuPtr<uint32_t> stopClk_d(BLK_SIZE);

        // warmup i-cache
        for(int i = 0; i < WARM_UP; i++){
            L2_TLB_detect_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
        }
        hipDeviceSynchronize();

        // 
        L2_TLB_detect_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
        hipDeviceSynchronize();

        HostPtr<uint32_t> startClk_h;
        HostPtr<uint32_t> stopClk_h;
        startClk_d.ToHostPtr(startClk_h);
        stopClk_d.ToHostPtr(stopClk_h);

        double clockCycles_avg = 0.0;
        for(int i = 0; i < BLK_SIZE; i++){
            // std::cout << startClk_h(i) << " " << stopClk_h(i) << std::endl;
            clockCycles_avg += stopClk_h(i) - startClk_h(i);
        }
        clockCycles_avg /= (BLK_SIZE * UNROLL);
        std::cout << "ARR_SIZE_IN_MB: " << (ARR_SIZE_IN_M * 8) << ", Global Mem access latency: " << clockCycles_avg << " cycles" << std::endl;
    }
}

int main(){
    // 理论上应该是
    // 一个元素8B 数组字节大小上限为8192 + 1024 = 9216MB 元素上限为1152M
    // 从64M个元素(512MB) 每次递增64M(512MB) 一直到1152M个(9216MB)

    // 但是阉割版的3060只有不到6G的全局内存 数组字节大小上限为4096 + 1024 = 5120MB 元素上限为640M
    // 4096MB / 8B = 512M ; 3072MB / 8B = 3*128M = 384M
    // 6144MB/8B = 3*256 = 768M 就out of memory了。至此没有 L2TLB miss的情况。
    for(uint32_t arr_size = 64; arr_size < 768; arr_size += 64){
        L2_TLB_detect_run(arr_size);
    }
}


// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L2_TLB_detect L2_TLB_detect.cu
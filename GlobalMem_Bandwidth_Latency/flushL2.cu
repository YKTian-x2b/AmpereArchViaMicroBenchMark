
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

const int L2_FLUSH_SIZE = (1 << 20) * 128;

template <int BLOCK>
__global__ void flush_l2_kernel(const int *x, int *y) {
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;

    const int *x_ptr = x + blockIdx.x * BLOCK + warp_id * 32;
    int sum = 0;

    #pragma unroll
    for (int i = 0; i < 32; ++i) {
        const int *ldg_ptr = x_ptr + (lane_id ^ i);

        asm volatile (
            "{.reg .s32 val;\n"
            " ld.global.cg.b32 val, [%1];\n"
            " add.s32 %0, val, %0;}\n"
            : "+r"(sum) : "l"(ldg_ptr)
        );
    }

    if (sum != 0) {
        *y = sum;
    }
}

void flush_l2() {
    int *x;
    int *y;
    hipMalloc(&x, L2_FLUSH_SIZE);
    hipMalloc(&y, sizeof(int));
    hipMemset(x, 0, L2_FLUSH_SIZE);

    int n = L2_FLUSH_SIZE / sizeof(int);
    flush_l2_kernel<128><<<n / 128, 128>>>(x, y);

    hipFree(x);
    hipFree(y);
}


int main(){
    flush_l2();
}


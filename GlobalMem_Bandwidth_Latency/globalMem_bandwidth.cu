
#include <hip/hip_runtime.h>
// Global Mem的理论访存峰值吞吐 = mem frequency*DDR_factor * 总线宽度 
// GDDR和DDR的区别在于 图形内存的有效带宽是DDR的4倍。 DDR就是双倍数据速率。
// 这里nvidia-smi 查出来的 mem freq 就是已经乘4了
// 3060     7000M * 2 * 192/8  = 336 GB/s
// turing t4  5000M * 2 * 256/8  = 320GB/s
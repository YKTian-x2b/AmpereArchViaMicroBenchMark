#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hostptr.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdio>

#define WARMUP_ITER 200
#define BENCH_ITER 200
#define WARP_SIZE 32
#define BLK_SIZE 512
// 希望是 （SM个数 * 一个SM能启动的ThreadBlock数） 90 的倍数 要超大才能形成有效流水线
#define GRD_SIZE 90000
#define UNROLL 16
// Global Mem size 不到 6 * 1024*1024*1024字节
const size_t N_DATA = 1024*1024*256;

__device__ __forceinline__
float ldg_cv(const void *ptr) {
    float ret;
    asm volatile (
        "ld.global.cv.f32 %0, [%1];"
        : "=f"(ret)
        : "l"(ptr)
    );
    return ret;
}

__global__ void globalMemBW_test_kernel(const float *Arr, float *out) {
    int offset = (BLK_SIZE * UNROLL * blockIdx.x + threadIdx.x) % N_DATA;
    const float *ldg_ptr = Arr + offset;
    float sum = 0;

    #pragma unroll
    for (int i = 0; i < UNROLL; ++i) {
        sum += ldg_cv(ldg_ptr + BLK_SIZE * i);
    }

    if(sum != 0){
        *out = sum;
    }
}

int main(){
    HostPtr<float> arr_h(N_DATA);
    arr_h.SetZeros();
    CuPtr<float> arr_d(arr_h);
    CuPtr<float> y_d(N_DATA);

    std::cout << "Warming Up ..." << std::endl;
    // warm up
    for (int i = 0; i < WARMUP_ITER; ++i) {
        globalMemBW_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(arr_d.GetPtr(), y_d.GetPtr());
    }

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float time_ms;
    std::cout << "Running ..." << std::endl;
    checkCudaErrors(hipEventRecord(start));
    for (int i = 0; i < BENCH_ITER ; ++i) {
        globalMemBW_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(arr_d.GetPtr(), y_d.GetPtr());
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&time_ms, start, stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    double gbps = ((double)((GRD_SIZE*BLK_SIZE*UNROLL) * sizeof(float)) / 1e9) /
                  ((double)time_ms / BENCH_ITER / 1e3);
    printf("Global Memory Bandwidth: %lfGB/s\n", gbps);
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_bandwidth globalMem_bandwidth.cu

// 驱动调试
// nvcc --keep --keep-dir midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_bandwidth globalMem_bandwidth.cu
// cuasm --bin2asm midRes/globalMem_bandwidth.sm_86.cubin -o midRes/globalMem_bandwidth.sm_86.cuasm
// rm midRes/*
// cuasm --asm2bin midRes/globalMem_bandwidth.sm_86.cuasm -o midRes/globalMem_bandwidth.sm_86.cubin 


// GRD_SIZE 9000 的结果
// Global Memory Bandwidth: 326.575679GB/s
// GRD_SIZE 90000 的结果
// Global Memory Bandwidth: 327.241499GB/s

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hostptr.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdio>

#define WARMUP_ITER 200
#define BENCH_ITER 200
#define WARP_SIZE 32
#define BLK_SIZE 512
// 希望是 （SM个数 * 一个SM能启动的ThreadBlock数） 90 的倍数 要超大才能形成有效流水线
#define GRD_SIZE 90000
#define UNROLL 16
// Global Mem size 不到 6 * 1024*1024*1024字节
const size_t N_DATA = 1024*1024*128;

__device__ __forceinline__
float2 ldg_cv_v2(const void *ptr) {
    float2 ret;
    asm volatile (
        "ld.global.cv.v2.f32 {%0, %1}, [%2];"
        : "=f"(ret.x), "=f"(ret.y)
        : "l"(ptr)
    );
    return ret;
}

__global__ void globalMemBW_test_kernel(const float2 *Arr, float *out) {
    int offset = (BLK_SIZE * UNROLL * blockIdx.x + threadIdx.x) % N_DATA;
    const float2 *ldg_ptr = Arr + offset;
    float sum = 0;

    #pragma unroll
    for (int i = 0; i < UNROLL; ++i) {
        float2 tmp = ldg_cv_v2(ldg_ptr + BLK_SIZE * i);
        sum += tmp.x + tmp.y;
    }

    if(sum != 0){
        *out = sum;
    }
}

int main(){
    HostPtr<float2> arr_h(N_DATA);
    arr_h.SetZeros();
    CuPtr<float2> arr_d(arr_h);
    CuPtr<float> out_d(N_DATA);

    std::cout << "Warming Up ..." << std::endl;
    // warm up
    for (int i = 0; i < WARMUP_ITER; ++i) {
        globalMemBW_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(arr_d.GetPtr(), out_d.GetPtr());
    }

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float time_ms;
    std::cout << "Running ..." << std::endl;
    checkCudaErrors(hipEventRecord(start));
    for (int i = 0; i < BENCH_ITER ; ++i) {
        globalMemBW_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(arr_d.GetPtr(), out_d.GetPtr());
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&time_ms, start, stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    double gbps = ((double)((GRD_SIZE*BLK_SIZE*UNROLL) * sizeof(float2)) / 1e9) /
                  ((double)time_ms / BENCH_ITER / 1e3);
    printf("Global Memory Bandwidth: %lfGB/s\n", gbps);
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_bandwidth_float2 globalMem_bandwidth_float2.cu

// 驱动调试
// nvcc --keep --keep-dir midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_bandwidth_float2 globalMem_bandwidth_float2.cu
// cuasm --bin2asm midRes/globalMem_bandwidth_float2.sm_86.cubin -o midRes/globalMem_bandwidth_float2.sm_86.cuasm

// cuasm --asm2bin midRes/globalMem_bandwidth_float2.sm_86.cuasm -o midRes/globalMem_bandwidth_float2.sm_86.cubin 


// GRD_SIZE 90000 的结果
// Global Memory Bandwidth: 321.044992GB/s

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hostptr.hpp"
#include "cuptr.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

#define GRD_SIZE 1
#define BLK_SIZE 32
#define UNROLL 10
// STRIDE 应该比 L2 cacheline 大，以避免l2缓存命中。
// Turing T4是64字节
#define STRIDE (1)

// workspace size in byte to flush L2 cache
const int L2_FLUSH_SIZE = (1 << 20) * 128;

template <int BLOCK>
__global__ void flush_l2_kernel(const int *x, int *y) {
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;

    const int *x_ptr = x + blockIdx.x * BLOCK + warp_id * 32;
    int sum = 0;

    #pragma unroll
    for (int i = 0; i < 32; ++i) {
        const int *ldg_ptr = x_ptr + (lane_id ^ i);

        asm volatile (
            "{.reg .s32 val;\n"
            " ld.global.cg.b32 val, [%1];\n"
            " add.s32 %0, val, %0;}\n"
            : "+r"(sum) : "l"(ldg_ptr)
        );
    }

    if (sum != 0) {
        *y = sum;
    }
}

void flush_l2() {
    int *x;
    int *y;
    hipMalloc(&x, L2_FLUSH_SIZE);
    hipMalloc(&y, sizeof(int));
    hipMemset(x, 0, L2_FLUSH_SIZE);

    int n = L2_FLUSH_SIZE / sizeof(int);
    flush_l2_kernel<128><<<n / 128, 128>>>(x, y);
    hipDeviceSynchronize();

    hipFree(x);
    hipFree(y);
}

__device__ __forceinline__
uint32_t ldg_cv(const void *ptr) {
    uint32_t ret;
    asm volatile (
        "ld.global.cv.b32 %0, [%1];"
        : "=r"(ret)
        : "l"(ptr)
    );
    return ret;
}

__device__ __forceinline__
uint32_t ldg_cg(const void *ptr) {
    uint32_t ret;
    asm volatile (
        "ld.global.cg.b32 %0, [%1];"
        : "=r"(ret)
        : "l"(ptr)
    );
    return ret;
}

__device__ __forceinline__
uint32_t ldg_ca(const void *ptr) {
    uint32_t ret;
    asm volatile (
        "ld.global.ca.b32 %0, [%1];"
        : "=r"(ret)
        : "l"(ptr)
    );
    return ret;
}

__global__ void gloMem_latency_test_kernel(uint32_t *startClk, uint32_t *stopClk, uint32_t *out, uint32_t* arr){
    int tid = threadIdx.x;
    uint32_t offset = tid;

    uint32_t start, stop;
    asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");

    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        offset = ldg_ca(arr+offset);
        // offset = ldg_cg(arr+offset);
        // offset = ldg_ca(arr+offset);
    }

    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid] = stop;

    if(offset == ~0x0){
        *out = offset;
    }
}

int main(){
    const uint32_t ARR_NUM = (UNROLL + 2) * STRIDE;
    HostPtr<uint32_t> arr_h(ARR_NUM);
    uint32_t *arr_ptr = arr_h.GetPtr();
    for(int i = 0; i < ARR_NUM; i++){
        arr_ptr[i] = i + STRIDE;
    }
    CuPtr<uint32_t> arr_d(arr_h);

    CuPtr<uint32_t> out_d(BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);

    // pupulate l0/l1 i-cache
    gloMem_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());

    // flush L2 cache
    flush_l2();

    // 
    gloMem_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();

    HostPtr<uint32_t> out_h;
    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);

    double clockCycles_avg = 0.0;
    for(int i = 0; i < BLK_SIZE; i++){
        std::cout << startClk_h(i) << " " << stopClk_h(i) << std::endl;
        clockCycles_avg += stopClk_h(i) - startClk_h(i);
    }
    clockCycles_avg /= BLK_SIZE;

    printf("global memory load latency: %lf cycles\n", clockCycles_avg / UNROLL);
}


// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_latency globalMem_latency.cu

// nvcc --keep --keep-dir latency_midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_latency globalMem_latency.cu
// cuasm --bin2asm latency_midRes/globalMem_latency.sm_86.cubin -o latency_midRes/globalMem_latency.sm_86.cuasm
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hostptr.hpp"
#include "cuptr.hpp"
#include "flushL2.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

#define GRD_SIZE 1
#define BLK_SIZE 32
#define UNROLL 50
// STRIDE 应该比 L2 cacheline 大，以避免l2缓存命中。同时应该保证数组能让BLK_SIZE个线程访问
// Turing T4是64字节
#define STRIDE 256

__device__ __forceinline__
uint32_t ldg_cv(const void *ptr) {
    uint32_t ret;
    asm volatile (
        "ld.global.cv.b32 %0, [%1];"
        : "=r"(ret)
        : "l"(ptr)
    );
    return ret;
}

__global__ void gloMem_latency_test_kernel(uint32_t *startClk, uint32_t *stopClk, uint32_t *out, uint32_t* arr){
    int tid = threadIdx.x;
    uint32_t offset = tid;

    offset = ldg_cv(arr+offset);

    uint32_t start, stop;
    asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");

    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        offset = ldg_cv(arr+offset);
    }

    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid] = stop;

    if(offset == ~0x0){
        *out = offset;
    }
}

int main(){
    const uint32_t ARR_NUM = (UNROLL + 2) * STRIDE;
    HostPtr<uint32_t> arr_h(ARR_NUM);
    uint32_t *arr_ptr = arr_h.GetPtr();
    for(int i = 0; i < ARR_NUM; i++){
        arr_ptr[i] = i + STRIDE;
    }
    CuPtr<uint32_t> arr_d(arr_h);

    CuPtr<uint32_t> out_d(BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);

    // pupulate l0/l1 i-cache
    gloMem_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());

    // flush L2 cache
    flush_l2();

    // 
    gloMem_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();

    HostPtr<uint32_t> out_h;
    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);

    double clockCycles_avg = 0.0;
    for(int i = 0; i < BLK_SIZE; i++){
        // std::cout << startClk_h(i) << " " << stopClk_h(i) << std::endl;
        clockCycles_avg += stopClk_h(i) - startClk_h(i);
    }
    clockCycles_avg /= BLK_SIZE;

    printf("global memory load latency: %lf cycles\n", clockCycles_avg / UNROLL);
}


// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_latency_IMAD globalMem_latency_IMAD.cu

// nvcc --keep --keep-dir latency_midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_latency_IMAD globalMem_latency_IMAD.cu
// cuasm --bin2asm latency_midRes/globalMem_latency_IMAD.sm_86.cubin -o latency_midRes/globalMem_latency_IMAD.sm_86.cuasm
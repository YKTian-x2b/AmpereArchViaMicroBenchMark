#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hostptr.hpp"
#include "cuptr.hpp"
#include "flushL2.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

#define GRD_SIZE 1
#define BLK_SIZE 32
// 【10, 50, 100]
#define UNROLL 50
// STRIDE 应该比 L2 cacheline 大，避免l2缓存命中。同时应该保证数组能让BLK_SIZE个线程访问
// Turing T4的cacheline是64字节
// latency会随着STRIDE的增大而增大
#define STRIDE 128

__device__ __forceinline__
void ldg_ca(void **&ldg_ptr) {
    asm volatile (
        "ld.global.ca.b64 %0, [%0];\n"
        : "+l"(ldg_ptr)
        : : "memory"
    );
}

__device__ __forceinline__
void ldg_cg(void **&ldg_ptr) {
    asm volatile (
        "ld.global.cg.b64 %0, [%0];\n"
        : "+l"(ldg_ptr)
        : : "memory"
    );
}

__global__ void gloMem_latency_test_kernel(uint32_t *startClk, uint32_t *stopClk, void **out, void **arr){
    int tid = threadIdx.x;
    void **ldg_ptr = arr +  tid;
    // populate L2 TLB
    ldg_cg(ldg_ptr);

    uint32_t start, stop;
    asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");

    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        ldg_cg(ldg_ptr);
    }

    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid] = stop;

    if(ldg_ptr == nullptr){
        *out = ldg_ptr;
    }
}

double doTest(){
    uint32_t ARR_SIZE = (UNROLL + 2) * STRIDE;

    CuPtr<void*> arr_d(ARR_SIZE);
    HostPtr<void*> arr_h(ARR_SIZE);
    void **h_ptr = arr_h.GetPtr();
    void **d_ptr = arr_d.GetPtr();
    for(int i = 0; i < ARR_SIZE; i++){
        uint32_t offset = (i + STRIDE) % ARR_SIZE;
        h_ptr[i] = (void*)(d_ptr + offset);
    }
    uint32_t arrBytes = ARR_SIZE * sizeof(void*);
    checkCudaErrors(hipMemcpy(d_ptr, h_ptr, arrBytes, hipMemcpyHostToDevice));

    CuPtr<void*> out_d(BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);

    // pupulate l0/l1 i-cache
    gloMem_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), d_ptr);

    // flush L2 cache
    flush_l2();

    // 
    gloMem_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), d_ptr);
    hipDeviceSynchronize();

    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);

    double clockCycles = stopClk_h(0) - startClk_h(0);
    return clockCycles/UNROLL;
}

int main(){
    // 因为结果不稳定 所以10次实验取均值
    int times = 10;
    double Avg_res = 0.0;
    for(int i = 0; i < times; i++){
        Avg_res += doTest();
    }
    printf("In the case of L2 miss and L2 TLB hit, global memory load latency is: %lf cycles\n", (Avg_res/times));
}


// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_latency_None globalMem_latency_None.cu

// nvcc --keep --keep-dir latency_midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_latency_None globalMem_latency_None.cu
// cuasm --bin2asm latency_midRes/globalMem_latency_None.sm_86.cubin -o latency_midRes/globalMem_latency_None.sm_86.cuasm
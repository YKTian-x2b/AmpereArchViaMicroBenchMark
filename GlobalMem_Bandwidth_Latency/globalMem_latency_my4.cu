#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hostptr.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

#define GRD_SIZE 1
#define BLK_SIZE 1
#define UNROLL 256

// workspace size in byte to flush L2 cache
const int L2_FLUSH_SIZE = (1 << 20) * 128;

template <int BLOCK>
__global__ void flush_l2_kernel(const int *x, int *y) {
    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;

    const int *x_ptr = x + blockIdx.x * BLOCK + warp_id * 32;
    int sum = 0;

    #pragma unroll
    for (int i = 0; i < 32; ++i) {
        const int *ldg_ptr = x_ptr + (lane_id ^ i);

        asm volatile (
            "{.reg .s32 val;\n"
            " ld.global.cg.b32 val, [%1];\n"
            " add.s32 %0, val, %0;}\n"
            : "+r"(sum) : "l"(ldg_ptr)
        );
    }

    if (sum != 0) {
        *y = sum;
    }
}

void flush_l2() {
    int *x;
    int *y;
    hipMalloc(&x, L2_FLUSH_SIZE);
    hipMalloc(&y, sizeof(int));
    hipMemset(x, 0, L2_FLUSH_SIZE);

    int n = L2_FLUSH_SIZE / sizeof(int);
    flush_l2_kernel<128><<<n / 128, 128>>>(x, y);
    hipDeviceSynchronize();

    hipFree(x);
    hipFree(y);
}

__device__ __forceinline__
uint32_t ldg_cv(const void *ptr) {
    uint32_t ret;
    asm volatile (
        "ld.global.cv.b32 %0, [%1];"
        : "=r"(ret)
        : "l"(ptr)
    );
    return ret;
}

__device__ __forceinline__
uint32_t ldg_cg(const void *ptr) {
    uint32_t ret;
    asm volatile (
        "ld.global.cg.b32 %0, [%1];"
        : "=r"(ret)
        : "l"(ptr)
    );
    return ret;
}

__device__ __forceinline__
uint32_t ldg_ca(const void *ptr) {
    uint32_t ret;
    asm volatile (
        "ld.global.ca.b32 %0, [%1];"
        : "=r"(ret)
        : "l"(ptr)
    );
    return ret;
}

__device__ __forceinline__
uint32_t ldg_nc(const void *ptr) {
    uint32_t ret;
    asm volatile (
        "ld.global.nc.b32 %0, [%1];"
        : "=r"(ret)
        : "l"(ptr)
    );
    return ret;
}


__global__ void gloMem_latency_test_kernel(uint32_t *elapsedTime, uint32_t *out, uint32_t *arr){
    __shared__ uint32_t s_tvalue[UNROLL];
    __shared__ uint32_t s_index[UNROLL];

    uint32_t sum = 0;
    uint32_t j = 0;
    // j= ldg_ca(arr+j);

    uint32_t start_time, stop_time;
    for(int it = 0; it < UNROLL; it++){
        
        asm volatile ("mov.u32 %0, %%clock;" :"=r"(start_time) :: "memory");

        j= ldg_ca(arr+j);
        sum += j;

        asm volatile ("mov.u32 %0, %%clock;" :"=r"(stop_time) :: "memory");
        
        s_tvalue[it] = stop_time - start_time;
    }

    for(int i = 0; i < UNROLL; i++){
        elapsedTime[i] = s_tvalue[i];
    }
    
    *out = sum;
}

void gloMem_latency_test_run(const uint32_t STRIDE, const uint32_t ARR_NUM){
    HostPtr<uint32_t> arr_h(ARR_NUM);
    uint32_t *arr_ptr = arr_h.GetPtr();
    for(uint32_t  i = 0; i < ARR_NUM; i++){
        arr_ptr[i] = (i + STRIDE) % ARR_NUM;
    }
    CuPtr<uint32_t> arr_d(arr_h);
    CuPtr<uint32_t> out_d(1);
    CuPtr<uint32_t> elapsedTime_d(UNROLL);

    gloMem_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(elapsedTime_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();

    flush_l2();

    gloMem_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(elapsedTime_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();

    // HostPtr<uint32_t> out_h;
    HostPtr<uint32_t> elapsedTime_h;
    elapsedTime_d.ToHostPtr(elapsedTime_h);
    // out_d.ToHostPtr(out_h);

    for(int i = 0; i < UNROLL; i++){
        std::cout << "elapsedTime_h(" << i << "): " << elapsedTime_h(i) << std::endl;
    }
}

void doTest(){
    const uint32_t stride = 1;
    const uint32_t arr_size = 512;
    gloMem_latency_test_run(stride, arr_size);
}

int main(){
    doTest();
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_latency_my4 globalMem_latency_my4.cu

// nvcc --keep --keep-dir latency_midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_latency_my4 globalMem_latency_my4.cu
// cuasm --bin2asm latency_midRes/globalMem_latency_my4.sm_86.cubin -o latency_midRes/globalMem_latency_my4.sm_86.cuasm
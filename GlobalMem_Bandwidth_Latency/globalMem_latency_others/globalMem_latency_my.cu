#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hostptr.hpp"
#include "cuptr.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstdio>

#define GRD_SIZE 1
#define BLK_SIZE 32
#define WARM_UP 100
#define UNROLL 10

__device__ __forceinline__
uint32_t ldg_cv(const void *ptr) {
    uint32_t ret;
    asm volatile (
        "ld.global.cv.b32 %0, [%1];"
        : "=r"(ret)
        : "l"(ptr)
    );
    return ret;
}

__device__ __forceinline__
uint32_t ldg_cg(const void *ptr) {
    uint32_t ret;
    asm volatile (
        "ld.global.cg.b32 %0, [%1];"
        : "=r"(ret)
        : "l"(ptr)
    );
    return ret;
}

__device__ __forceinline__
uint32_t ldg_ca(const void *ptr) {
    uint32_t ret;
    asm volatile (
        "ld.global.ca.b32 %0, [%1];"
        : "=r"(ret)
        : "l"(ptr)
    );
    return ret;
}

__device__ __forceinline__
uint32_t ldg_nc(const void *ptr) {
    uint32_t ret;
    asm volatile (
        "ld.global.nc.b32 %0, [%1];"
        : "=r"(ret)
        : "l"(ptr)
    );
    return ret;
}


__global__ void gloMem_latency_test_kernel(uint32_t *startClk, uint32_t *stopClk, uint32_t *out, uint32_t* arr){
    int tid = threadIdx.x;
    uint32_t offset = tid;
    // uint32_t *lg_ptr = arr+offset;

    uint32_t start, stop;
    asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");

    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        // offset = ldg_cv(arr+offset);
        // offset = ldg_cg(arr+offset);
        // offset = ldg_ca(arr+offset);
        offset = ldg_nc(arr+offset);

        // asm volatile (
        //     "ld.global.ca.b32 %0, [%1];"
        //     : "=r"(offset)
        //     : "l"(lg_ptr)
        // );
        // lg_ptr += offset;

        // offset = arr[offset];
    }

    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid] = stop;

    if(offset == ~0x0){
        *out = offset;
    }
}

void gloMem_latency_test_run(uint32_t STRIDE){
    const uint32_t ARR_NUM = (UNROLL + 2) * STRIDE;
    HostPtr<uint32_t> arr_h(ARR_NUM);
    uint32_t *arr_ptr = arr_h.GetPtr();
    for(int i = 0; i < ARR_NUM; i++){
        arr_ptr[i] = i + STRIDE;
    }
    CuPtr<uint32_t> arr_d(arr_h);

    CuPtr<uint32_t> out_d(BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);

    // // warmup i-cache
    // for(int i = 0; i < WARM_UP; i++){
    //     gloMem_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    // }
    // hipDeviceSynchronize();

    // 
    gloMem_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();

    HostPtr<uint32_t> out_h;
    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);

    double clockCycles_avg = 0.0;
    for(int i = 0; i < BLK_SIZE; i++){
        // std::cout << startClk_h(i) << " " << stopClk_h(i) << std::endl;
        clockCycles_avg += stopClk_h(i) - startClk_h(i);
    }
    clockCycles_avg /= (BLK_SIZE * UNROLL);
    std::cout << "stride: " << STRIDE << ", global memory load latency: " << clockCycles_avg << " cycles" << std::endl;
    // // 不能用
    // std::string path("res/globalMem_latency_res1.txt");
    // std::ofstream fout(path); 
	// std::streambuf* pOld = std::cout.rdbuf(fout.rdbuf());
}

void doTest(){
    // for(int i = 256*256; i < 1024*1024*4; i += 256*256){
    //     gloMem_latency_test_run(i);
    // }

    // for(int i = 1; i < 128; i++){
    //     gloMem_latency_test_run(i);
    // }
    
    gloMem_latency_test_run(1024*256);
    // gloMem_latency_test_run(2*1024*256);
    // gloMem_latency_test_run(3*1024*256);
    // gloMem_latency_test_run(4*1024*256);
}

int main(){
    doTest();
}


// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_latency_my globalMem_latency_my.cu

// nvcc --keep --keep-dir latency_midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_latency_my globalMem_latency_my.cu
// cuasm --bin2asm latency_midRes/globalMem_latency_my.sm_86.cubin -o latency_midRes/globalMem_latency_my.sm_86.cuasm
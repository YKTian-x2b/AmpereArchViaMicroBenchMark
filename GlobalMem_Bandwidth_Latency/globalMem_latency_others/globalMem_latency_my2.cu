#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hostptr.hpp"
#include "cuptr.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <cstdio>

#define GRD_SIZE 1
#define BLK_SIZE 32
#define WARM_UP 100
#define UNROLL 10

__device__ __forceinline__
uint32_t ldg_cv(const void *ptr) {
    uint32_t ret;
    asm volatile (
        "ld.global.cv.b32 %0, [%1];"
        : "=r"(ret)
        : "l"(ptr)
    );
    return ret;
}

__device__ __forceinline__
uint32_t ldg_cg(const void *ptr) {
    uint32_t ret;
    asm volatile (
        "ld.global.cg.b32 %0, [%1];"
        : "=r"(ret)
        : "l"(ptr)
    );
    return ret;
}

__device__ __forceinline__
uint32_t ldg_ca(const void *ptr) {
    uint32_t ret;
    asm volatile (
        "ld.global.ca.b32 %0, [%1];"
        : "=r"(ret)
        : "l"(ptr)
    );
    return ret;
}

__device__ __forceinline__
uint32_t ldg_nc(const void *ptr) {
    uint32_t ret;
    asm volatile (
        "ld.global.nc.b32 %0, [%1];"
        : "=r"(ret)
        : "l"(ptr)
    );
    return ret;
}


__global__ void gloMem_latency_test_kernel(uint32_t *startClk, uint32_t *stopClk, uint32_t *out, uint32_t* arr){
    int tid = threadIdx.x;
    uint32_t offset = tid;
    // uint32_t *lg_ptr = arr+offset;

    uint32_t start, stop;
    asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");

    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        // offset = ldg_cv(arr+offset);
        // offset = ldg_cg(arr+offset);
        offset = ldg_ca(arr+offset);
        // offset = ldg_nc(arr+offset);

        // asm volatile (
        //     "ld.global.ca.b32 %0, [%1];"
        //     : "=r"(offset)
        //     : "l"(lg_ptr)
        // );
        // lg_ptr += offset;

        // offset = arr[offset];
    }

    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid] = stop;

    if(offset == ~0x0){
        *out = offset;
    }
}

float gloMem_latency_test_run(uint32_t STRIDE, uint32_t ARR_SIZE_IN_MB){
    const unsigned long long ARR_SIZE = ARR_SIZE_IN_MB * 1024 * 1024;
    const unsigned long long ARR_NUM = ARR_SIZE / sizeof(uint32_t);
    HostPtr<uint32_t> arr_h(ARR_NUM);
    uint32_t *arr_ptr = arr_h.GetPtr();
    for(unsigned long long  i = 0; i < ARR_NUM; i++){
        arr_ptr[i] = (i + STRIDE) % ARR_NUM;
    }
    CuPtr<uint32_t> arr_d(arr_h);

    CuPtr<uint32_t> out_d(BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);

    // warmup i-cache
    gloMem_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();

    // 
    gloMem_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();

    HostPtr<uint32_t> out_h;
    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);

    double clockCycles_avg = 0.0;
    for(int i = 0; i < BLK_SIZE; i++){
        // std::cout << startClk_h(i) << " " << stopClk_h(i) << std::endl;
        clockCycles_avg += stopClk_h(i) - startClk_h(i);
    }
    clockCycles_avg /= (BLK_SIZE * UNROLL);
    return clockCycles_avg;
}

void doTest(){
    // for(int i = 256*256; i < 1024*1024*4; i += 256*256){
    //     double clockCycles_avg = gloMem_latency_test_run(i);
    //     std::cout << "stride: " << i << ", global memory load latency: " << clockCycles_avg << " cycles" << std::endl;
    // }

    // for(int i = 1; i < 128; i++){
    //     double clockCycles_avg = gloMem_latency_test_run(i);
    //     std::cout << "stride: " << i << ", global memory load latency: " << clockCycles_avg << " cycles" << std::endl;
    // }

    /*
    const uint32_t STRIDE = 8 * 1024 * 1024;    // 32MB 再试试16MB
    const uint32_t arr_size = 3192;
    double clockCycles_avg = gloMem_latency_test_run(STRIDE, arr_size);
    std::cout << "arr_size: " << arr_size << "MB, global memory load latency: " << clockCycles_avg << " cycles" << std::endl;
    // for(int arr_size = 512; arr_size < 4096; arr_size += 512){
    //     double clockCycles_avg = gloMem_latency_test_run(STRIDE, arr_size);
    //     std::cout << "arr_size: " << arr_size << ", global memory load latency: " << clockCycles_avg << " cycles" << std::endl;
    // }
    */

    const uint32_t STRIDE = 512 * 1024;    // 32MB 再试试16MB
    for(int arr_size = 4; arr_size < 512; arr_size += 4){
        double clockCycles_avg = gloMem_latency_test_run(STRIDE, arr_size);
        std::cout << "arr_size: " << arr_size << ", global memory load latency: " << clockCycles_avg << " cycles" << std::endl;
    }
    // double clockCycles_avg = gloMem_latency_test_run(STRIDE, arr_size);
    // std::cout << "arr_size: " << arr_size << "MB, global memory load latency: " << clockCycles_avg << " cycles" << std::endl;

}

int main(){
    doTest();
}


// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_latency_my2 globalMem_latency_my2.cu

// nvcc --keep --keep-dir latency_midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_latency_my2 globalMem_latency_my2.cu
// cuasm --bin2asm latency_midRes/globalMem_latency_my2.sm_86.cubin -o latency_midRes/globalMem_latency_my2.sm_86.cuasm
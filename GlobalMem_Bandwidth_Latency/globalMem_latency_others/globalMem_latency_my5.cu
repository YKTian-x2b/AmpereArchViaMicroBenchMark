#include "hip/hip_runtime.h"
#include "cuptr.hpp"
#include "hostptr.hpp"
#include "hip/hip_runtime_api.h"
#include "flushL2.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

#define GRD_SIZE 1
#define BLK_SIZE 32
#define UNROLL 10
#define WARM_UP 100
#define STRIDE 1

__device__ __forceinline__
void ldg_cv(void **&ldg_ptr) {
    asm volatile (
        "ld.global.cv.b64 %0, [%0];\n"
        : "+l"(ldg_ptr)
        : : "memory"
    );
}

__device__ __forceinline__
void ldg_cg(void **&ldg_ptr) {
    asm volatile (
        "ld.global.cg.b64 %0, [%0];\n"
        : "+l"(ldg_ptr)
        : : "memory"
    );
}

__device__ __forceinline__
void ldg_ca(void **&ldg_ptr) {
    asm volatile (
        "ld.global.ca.b64 %0, [%0];\n"
        : "+l"(ldg_ptr)
        : : "memory"
    );
}


__global__ void gloMem_latency_test_kernel(uint32_t *startClk, uint32_t *stopClk, void **out, void **arr){
    int tid = threadIdx.x;
    void ** ldg_ptr = arr + tid;

    // pupulate L2 TLB
    ldg_ca(ldg_ptr);

    uint32_t start, stop;
    asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");

    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        ldg_ca(ldg_ptr);
    }  
    
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid] = stop;

    if(ldg_ptr == nullptr){
        *out = ldg_ptr;
    }
}

int main(){
    // 得保证ARR_SIZE至少是BLK_SIZE 设置最小为128个元素
    const uint32_t min_arr_size = 128;
    const uint32_t ARR_SIZE = max(min_arr_size, (UNROLL + 2) * STRIDE);
    CuPtr<void*> arr_d(ARR_SIZE);
    HostPtr<void*> arr_h(ARR_SIZE);
    void **h_ptr = arr_h.GetPtr();
    void **d_ptr = arr_d.GetPtr();
    for(int i = 0; i < ARR_SIZE; i++){
        uint32_t offset = (i + STRIDE) % ARR_SIZE;
        h_ptr[i] = (void*)(d_ptr + offset);
    }
    uint32_t arrBytes = ARR_SIZE * sizeof(void*);
    checkCudaErrors(hipMemcpy(d_ptr, h_ptr, arrBytes, hipMemcpyHostToDevice));

    CuPtr<void*> out_d(BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);

    // warmup i-cache
    for(int i = 0; i < WARM_UP; i++){
        gloMem_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    }
    hipDeviceSynchronize();

    // flush_l2();

    // 
    gloMem_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();

    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);

    double clockCycles_avg = 0.0;
    for(int i = 0; i < BLK_SIZE; i++){
        // std::cout << startClk_h(i) << " " << stopClk_h(i) << std::endl;
        clockCycles_avg += stopClk_h(i) - startClk_h(i);
    }
    clockCycles_avg /= (BLK_SIZE * UNROLL);
    std::cout << "Global Memory access latency: " << clockCycles_avg << " cycles" << std::endl;
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_latency_my5 globalMem_latency_my5.cu

// nvcc --keep --keep-dir latency_midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/globalMem_latency_my5 globalMem_latency_my5.cu
// cuasm --bin2asm latency_midRes/globalMem_latency_my5.sm_86.cubin -o latency_midRes/globalMem_latency_my5.sm_86.cuasm
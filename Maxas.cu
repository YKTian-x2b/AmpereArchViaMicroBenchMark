#include "hip/hip_runtime.h"
#include "../Common.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <ctime>
#include "hipblas.h"

#include <hip/device_functions.h>
#include <>
#include <hip/hip_texture_types.h>
#include <>

// 四元向量加载 要求数据是 4 word alignment  这至少要求lda ldb是被四整除的 
// 缓存性能由L2 主导
typedef texture<float4, hipTextureType1D, hipReadModeElementType> floatTex;

floatTex  texA(0, hipFilterModePoint, hipAddressModeBorder);
floatTex  texB(0, hipFilterModePoint, hipAddressModeBorder);

// track registers用来持有纹理加载索引 这将削减主循环里半数的整数加法指令数量 提高FFMA和非FFMA指令的比例 这是使用纹理内存的主要优点
// 还需要额外4个单独的track变量来避免纹理加载间的 Idx*2 依赖屏障 这不会产生额外的IADD指令，只需要3个额外的寄存器
// 另外 纹理内存的使用让我们无需担心全局内存加载的越界问题，代码更简单；而且同样的代码还能加载半精度和更小精度的应用（!!!)

// STS.128指令会造成bank冲突，这不可避免且不会影响性能，因为 批加载和向量存储是 a net win
// bank冲突导致的指令reply可能与FFMA 双发

// 4 vec4 4 byte loads per thread per loop 
__global__ void sgemm_kernel_64(float *C,
	const int m,   const int n,   const int k,
	const int lda, const int ldb, const int ldc,
	float alpha, int *D)
{
    int tid = threadIdx.x;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    // 前32个线程处理A 后32个线程处理B
    // 列存A的lda是x维 行存B的ldb是y维
    int blk = tid >= 32 ? by : bx;
    int ldx = tid >= 32 ? ldb/4 : lda/4;
    floatTex tex = tid >= 32 ? texB : texA;

    // warpid 0前1后
    int tid2 = (tid>>4)&1;
    // laneid
    int tid15 = tid & 15;
    // tex的索引
    int track0 = blk*64/4 + tid15 + (ldx * tid2);
    int track2 = track0 + ldx * 2;
    int track4 = track0 + ldx * 4;
    int track6 = track0 + ldx * 6;
    // 
    int end = track0 + (k-8)*ldx;
    // write Shared
    int writeS = tid15*4*4 + tid2*64*4;
    int writeS += tid >= 32 ? 2048 : 0;

    while(track0 < end){
        // load tex store shared
        tex.1d.v4.f32.s32 loadX0, [tex, track0];
        tex.1d.v4.f32.s32 loadX2, [tex, track2];
        tex.1d.v4.f32.s32 loadX4, [tex, track4];
        tex.1d.v4.f32.s32 loadX6, [tex, track6];

        st.shared.v4.f32 [writeS + 4*0*64], loadX0;
        st.shared.v4.f32 [writeS + 4*2*64], loadX2;
        st.shared.v4.f32 [writeS + 4*4*64], loadX4;
        st.shared.v4.f32 [writeS + 4*6*64], loadX6;

        bar.sync 0;

        track0 += ldx * 8;
        track2 += ldx * 8;
        track4 += ldx * 8;
        track6 += ldx * 8;
        // 切buffer
        writeS ^= 4*16*64;
    }

    // 1111,1111 ~ 0000,0000   ----,xyz- => 0xyz,0000
    int readAs = ((tid >> 1) & 7) << 4;
    // ((tid & 0x30) >> 3):  0011,0000 --ab,---c => 00ab,0000 => 0000,0ab0 
    // (((tid & 0x30) >> 3) | (tid & 1)):  --ab,---c => 0000,000c => 0000,0abc 
    // (((tid & 0x30) >> 3) | (tid & 1)) << 4:  0abc,0000
    // +2048:  1000,0000,0000 => 1000,0abc,0000
    int readBs = (((tid & 0x30) >> 3) | (tid & 1)) << 4 + 2048;
    // --ab,xyzc readAs=0xyz,0000 readBs=1000,0abc,0000

    while(track0 < end){
        for(j = 0; j < 8; j++){
            // 计算当前行的时候预取后一行 最后一行换到第一行
            int prefetch = (j+1) % 8;
            // 双缓冲用奇偶行来实现
            if(j & 1){
                ld.shared.v4.f32 j0Ax00, [readAs + 4*(prefetch*64+0)];
                ld.shared.v4.f32 j0By00, [readBs + 4*(prefetch*64+0)];
                ld.shared.v4.f32 j0Ax32, [readAs + 4*(prefetch*64+32)];
                ld.shared.v4.f32 j0By32, [readBs + 4*(prefetch*64+32)];
            }
            else{
                ld.shared.v4.f32 j1Ax00, [readAs + 4*(prefetch*64+0)];
                ld.shared.v4.f32 j1By00, [readBs + 4*(prefetch*64+0)];
                ld.shared.v4.f32 j1Ax32, [readAs + 4*(prefetch*64+32)];
                ld.shared.v4.f32 j1By32, [readBs + 4*(prefetch*64+32)];
            }
        }
        // 2048
        readAs ^= 4*16*64;
        readBs ^= 4*16*64;
    }
    // 操作数重用缓存的每个源操作数槽有8个字节的数据。像FFMA这样的指令有3个源操作数插槽。
    // 每次发出指令时，都有一个标志可用于指定是否将再次使用每个操作数。
    // 因此，在同一操作数插槽中使用相同寄存器的下一条指令将不必转到寄存器库来获取其值。使用此功能，您可以看到如何避免寄存器bank冲突
    // 通过显式选择寄存器，可以通过operand reuse cache来避免register bank conflict
    // ptxas在避免reg bank conflict方面做得很好但不够完美，涉及向量指令的时候做的很差

    // 0-63 to be the C registers
    // 64-71 and 80-87 to be the double buffered blocking registers for matrix A
    // 72-79 and 88-95 to be the double buffered blocking registers for matrix B
    
}


int main(){
    
    dim3 blockSize(64);
    dim3 gridSize(Y/64, X/64);
    sgemm_kernel_64<<<gridSize, blockSize>>>();
}
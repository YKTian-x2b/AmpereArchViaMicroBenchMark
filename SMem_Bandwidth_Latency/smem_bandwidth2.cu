#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hostptr.hpp"
#include "cuptr.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdio>

#define GRD_SIZE 1
#define BLK_SIZE 1024
// 一个Block的共享内存上限是48KB 1<<15是32KB
#define SMEM_SIZE (1<<11)
#define WARP_SIZE 32
#define WARMUP 100


__global__ void SMem_writeBW_test_kernel(uint32_t *startClk, uint32_t *stopClk, float *out){
    int tid = threadIdx.x;
    __shared__ float4 Arr[SMEM_SIZE+BLK_SIZE]; // 很极限
    float4 reg = make_float4(tid+1.0, tid+2.0, tid+3.0, tid+4.0);

    uint32_t start = 0;
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");

    for(int i = 0; i < SMEM_SIZE; i += WARP_SIZE){
        Arr[tid + i] = reg;
    }

    uint32_t stop = 0;
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");
    asm volatile ("bar.sync 0;");

    startClk[tid] = start;
    stopClk[tid]= stop;
    float tmp = ((float*)Arr)[tid];
    if(tmp < 0){
        *out = tmp;
    }
}

int main(){
    CuPtr<float> out_d(SMEM_SIZE + BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);

    for (int i = 0; i < WARMUP; ++i) {
        SMem_writeBW_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr());
    }
    hipDeviceSynchronize();

    SMem_writeBW_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr());
    hipDeviceSynchronize();
   
    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    HostPtr<float> out_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);
    out_d.ToHostPtr(out_h);

    float clockCycles_avg = 0.0;
    float out_total = 0.0;
    for(int i = 0; i < BLK_SIZE; i++){
        clockCycles_avg += stopClk_h(i) - startClk_h(i);
        // std::cout << h_startClk[i] << " " << h_stopClk[i] << std::endl;
        out_total += out_h(i);
    }
    clockCycles_avg /= BLK_SIZE;

    std::cout << "clockCycles_avg: " << clockCycles_avg << std::endl;
    std::cout << "out_total: " << out_total << std::endl;
    // Block加载了全部的L1 cache Data
    uint32_t BytesAll = SMEM_SIZE * sizeof(float4) * (BLK_SIZE/WARP_SIZE);  // * (BLK_SIZE/WARP_SIZE);   //  * BLK_SIZE;
    //
    float BpCpSM = BytesAll / clockCycles_avg;
    std::cout << "Shared Mem Bandwidth is  " << BpCpSM <<  " Bytes per cycle per SM" << std::endl;
    std::cout << "Shared Mem Bandwidth is  " << BpCpSM * 1.7 <<  " GBps per SM" << std::endl;
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/smem_bandwidth2 smem_bandwidth2.cu

// nvcc --keep --keep-dir midRes_2 -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/smem_bandwidth2 smem_bandwidth2.cu
// cuasm --bin2asm midRes_2/smem_bandwidth2.sm_86.cubin -o midRes_2/smem_bandwidth2.sm_86.cuasm
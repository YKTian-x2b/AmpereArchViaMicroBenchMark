#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hostptr.hpp"
#include "cuptr.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdio>

#define GRD_SIZE 1
#define BLK_SIZE 1024
// 一个Block的共享内存上限是48KB 1<<15是32KB
#define SMEM_SIZE (1<<11)
#define WARP_SIZE 32
#define WARMUP 100


__global__ void SMem_writeBW_test_kernel(uint32_t *startClk, uint32_t *stopClk, float *out){
    int tid = threadIdx.x;
    __shared__ float Arr[SMEM_SIZE+BLK_SIZE];
    float reg = tid+1.0;

    uint32_t start, stop;
    asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");

    for(int i = 0; i < SMEM_SIZE; i += WARP_SIZE){
        Arr[tid + i] = reg;
    }

    asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");
    

    startClk[tid] = start;
    stopClk[tid]= stop;
    float tmp = Arr[tid];
    if(tmp == 0){
        *out = tmp;
    }
}

int main(){
    CuPtr<float> out_d(SMEM_SIZE + BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);

    // pupulate l0/l1 i-cache
    for (int i = 0; i < WARMUP; ++i) {
        SMem_writeBW_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr());
    }
    hipDeviceSynchronize();

    SMem_writeBW_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr());
    hipDeviceSynchronize();
   
    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    HostPtr<float> out_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);
    out_d.ToHostPtr(out_h);

    float clockCycles_avg = 0.0;
    float out_total = 0.0;
    for(int i = 0; i < BLK_SIZE; i++){
        clockCycles_avg += stopClk_h(i) - startClk_h(i);
        // std::cout << h_startClk[i] << " " << h_stopClk[i] << std::endl;
        out_total += out_h(i);
    }
    clockCycles_avg /= BLK_SIZE;

    std::cout << "clockCycles_avg: " << clockCycles_avg << std::endl;
    std::cout << "out_total: " << out_total << std::endl;
    // Block加载了全部的L1 cache Data
    uint32_t BytesAll = SMEM_SIZE * sizeof(float) * (BLK_SIZE/WARP_SIZE);  // * (BLK_SIZE/WARP_SIZE);   //  * BLK_SIZE;
    // 理论上限是64B 16个LD_ST单元 * 4B/LD_ST单元/周期 实测结果是 65B左右
    float BpCpSM = BytesAll / clockCycles_avg;
    std::cout << "Shared Mem Bandwidth is  " << BpCpSM <<  " Bytes per cycle per SM" << std::endl;
    std::cout << "Shared Mem Bandwidth is  " << BpCpSM * 1.7 <<  " GBps per SM" << std::endl;
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/smem_bandwidth_float smem_bandwidth_float.cu

// nvcc --keep --keep-dir bw_midRes_float -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/smem_bandwidth_float smem_bandwidth_float.cu
// cuasm --bin2asm bw_midRes_float/smem_bandwidth_float.sm_86.cubin -o bw_midRes_float/smem_bandwidth_float.sm_86.cuasm

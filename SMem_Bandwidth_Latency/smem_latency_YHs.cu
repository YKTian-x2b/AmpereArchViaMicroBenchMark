
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>

const int WARMUP = 100;
// number of LDS instructions to be timed
const int ROUND = 50;

__global__ __launch_bounds__(16, 1)
void smem_latency_kernel(const uint32_t *addr, uint32_t *ret, uint32_t *clk) {
    __shared__ uint32_t smem[16];

    smem[threadIdx.x] = addr[threadIdx.x];

    uint32_t start;
    uint32_t stop;
    uint32_t smem_addr;

    asm volatile (
        "{.reg .u64 u64addr;\n"
        " cvta.to.shared.u64 u64addr, %1;\n"
        " cvt.u32.u64 %0, u64addr;}\n"
        : "=r"(smem_addr)
        : "l"(smem + threadIdx.x)
    );

    asm volatile (
        "bar.sync 0;\n"
        "mov.u32 %0, %%clock;\n"
        : "=r"(start) : : "memory"
    );

    #pragma unroll
    for (int i = 0; i < ROUND; ++i) {
        /*
         * dependent LDS instructions to make sure that
         * LDS latency can not be hidden by parallel LDS.
         */
        asm volatile (
            "ld.shared.b32 %0, [%0];\n"
            : "+r"(smem_addr) : : "memory"
        );
    }

    asm volatile (
        "bar.sync 0;\n"
        "mov.u32 %0, %%clock;\n"
        : "=r"(stop) : : "memory"
    );

    clk[threadIdx.x] = stop - start;

    // dummy write back
    if (smem_addr == ~0x0) {
        *ret = smem_addr;
    }
}

int main() {
    uint32_t *h_addr;
    hipHostMalloc(&h_addr, 16 * sizeof(uint32_t), hipHostMallocDefault);

    for (int i = 0; i < 16; ++i) {
        h_addr[i] = i * sizeof(uint32_t);
    }

    uint32_t *d_addr, *d_ret;
    hipMalloc(&d_addr, 16 * sizeof(uint32_t));
    hipMalloc(&d_ret, sizeof(uint32_t));
    hipMemcpy(d_addr, h_addr, 16 * sizeof(uint32_t), hipMemcpyHostToDevice);

    uint32_t *d_clk;
    hipMalloc(&d_clk, 16 * sizeof(uint32_t));

    // pupulate l0/l1 i-cache
    for (int i = 0; i < WARMUP; ++i) {
        smem_latency_kernel<<<1, 16>>>(d_addr, d_ret, d_clk);
    }

    // shared memory latency benchmark
    smem_latency_kernel<<<1, 16>>>(d_addr, d_ret, d_clk);

    uint32_t h_clk[16];
    hipMemcpy(h_clk, d_clk, 16 * sizeof(uint32_t), hipMemcpyDeviceToHost);
    printf("shared memory latency %u cycles\n", h_clk[0] / ROUND);

    hipFree(d_addr);
    hipFree(d_ret);
    hipFree(d_clk);
    hipHostFree(h_addr);

    return 0;
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/smem_latency_YHs smem_latency_YHs.cu

// nvcc --keep --keep-dir latency_midRes_YHs -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/smem_latency_YHs smem_latency_YHs.cu
// cuasm --bin2asm latency_midRes_YHs/smem_latency_YHs.sm_86.cubin -o latency_midRes_YHs/smem_latency_YHs.sm_86.cuasm

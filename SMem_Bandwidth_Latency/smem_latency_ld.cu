#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hostptr.hpp"

#include <iostream>
#include <cstdio>
#include <cstdint>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GRD_SIZE 1
// 这里BLK_SIZE取32,结果也相同，我感觉跟 L1Cache和SharedMemory 取数据的方式有关系，一个按bank，一个按cacheline。
#define BLK_SIZE 16
#define UNROLL 50
#define WARMUP 100

// SMem(ld/st): 23/19 Cycles
__global__ void smem_ld_latency_test_kernel(uint32_t *startClk, uint32_t *stopClk, uint32_t *out){
    __shared__ uint32_t Arr[BLK_SIZE];
    int tid = threadIdx.x;
    uint32_t start, stop;
    
    // Arr[tid] = ((tid + BLK_SIZE/2) % BLK_SIZE) * sizeof(uint32_t);
    Arr[tid] = tid * sizeof(uint32_t);

    uint32_t arr_addr;
    // 之所以能64位强转32位，是因为共享内存是靠偏移量访问的（多次共享内存的分配是连续的）。
    // 声明一个64位寄存器u64addr，将通用地址转换为指向共享内存空间的指针并赋值给该寄存器，截断高位。
    asm volatile (
        "{.reg .u64 u64addr;\n"
        " cvta.to.shared.u64 u64addr, %1;\n"
        " cvt.u32.u64 %0, u64addr;}\n"
        : "=r"(arr_addr)
        : "l"(Arr + tid)
    );

    asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");
    
    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        asm volatile (
            "ld.shared.b32 %0, [%0];\n"
            : "+r"(arr_addr) : : "memory"
        );
    }
 
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid] = stop;
    out[tid] = arr_addr;
}

void doLoadTest(){
    CuPtr<uint32_t> out_d(BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);
    // warmup
    for (int i = 0; i < WARMUP; i++) {
        smem_ld_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr());
    }
    hipDeviceSynchronize();
    // run
    smem_ld_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr());
    hipDeviceSynchronize();

    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);

    double clockCycles = stopClk_h(0) - startClk_h(0);
    clockCycles /= UNROLL;

    printf("shared memory load latency: %lf cycles\n", clockCycles);
}

int main(){
    doLoadTest();
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/smem_latency_ld smem_latency_ld.cu

// nvcc --keep --keep-dir latency_midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/smem_latency_ld smem_latency_ld.cu
// cuasm --bin2asm latency_midRes/smem_latency_ld.sm_86.cubin -o latency_midRes/smem_latency_ld.sm_86.cuasm


// UNROLL500次的ld结果是：22.9620 约等于 23 个时钟周期
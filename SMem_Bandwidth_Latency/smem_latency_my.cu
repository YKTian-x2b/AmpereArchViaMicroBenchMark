#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hostptr.hpp"

#include <iostream>
#include <cstdio>
#include <cstdint>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GRD_SIZE 1
#define BLK_SIZE 32
#define SMEM_SIZE 32
#define UNROLL 50
#define WARMUP 100

// SMem(ld/st): 23/19 Cycles
__global__ __launch_bounds__(32, 1)
void smem_ld_latency_test_kernel(uint32_t *startClk, uint32_t *stopClk, uint32_t *out){
    __shared__ uint32_t Arr[SMEM_SIZE];
    int tid = threadIdx.x;
    uint32_t start, stop;
    
    // Arr[tid] = ((tid + SMEM_SIZE/2) % SMEM_SIZE) * sizeof(uint32_t);
    Arr[tid] = tid * sizeof(uint32_t);

    uint32_t arr_addr;
    asm volatile (
        "{.reg .u64 u64addr;\n"
        " cvta.to.shared.u64 u64addr, %1;\n"
        " cvt.u32.u64 %0, u64addr;}\n"
        : "=r"(arr_addr)
        : "l"(Arr + tid)
    );
    asm volatile ("bar.sync 0;");

    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");
    
    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        asm volatile (
            "ld.shared.b32 %0, [%0];\n"
            : "+r"(arr_addr) : : "memory"
        );
    }

    // 这儿可能可以加个同步
    // asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid] = stop;

    if(arr_addr == ~0x0){
        *out = arr_addr;
    }
}

__global__ __launch_bounds__(32, 1)
void smem_st_latency_test_kernel(uint32_t *startClk, uint32_t *stopClk, uint32_t *out){
    __shared__ uint32_t Arr[SMEM_SIZE+UNROLL];
    int tid = threadIdx.x;
    uint32_t start, stop;

    Arr[tid] = tid * sizeof(uint32_t);

    uint32_t arr_addr;
    asm volatile (
        "{.reg .u64 u64addr;\n"
        " cvta.to.shared.u64 u64addr, %1;\n"
        " cvt.u32.u64 %0, u64addr;}\n"
        : "=r"(arr_addr)
        : "l"(Arr + tid)
    );

    asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");
    
    // #pragma unroll
    // for(int i = 0; i < UNROLL; i++){
    //     asm volatile (
    //         "st.shared.b32 [%0], %1;\n"
    //         : "=r"(arr_addr) : "r"(tid) : "memory"
    //     );
    // }

    asm volatile (
        "st.shared.b32 [%0], %1;\n"
        : "=r"(arr_addr) : "r"(tid) : "memory"
    );

    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid] = stop;

    if(arr_addr == ~0x0){
        *out = Arr[tid];
    }
}

void smem_st_latency_test_run_drv(uint32_t *startClk, uint32_t *stopClk, uint32_t *out){
    static hipModule_t cuModule;
    static hipFunction_t kernel;
    static bool isInitialized = false;

    if (!isInitialized)
    {
        hipInit(0);

        // Create module from binary file
        hipModuleLoad(&cuModule, "latency_midRes/smem_latency_my.sm_86.cubin");

        // Get function handle from module 
        hipModuleGetFunction(&kernel, cuModule, "_Z27smem_st_latency_test_kernelPjS_S_");

        isInitialized = true;
    }

    void* args[] = { (void*)&startClk, (void*)&stopClk, (void*)&out };
    hipModuleLaunchKernel(kernel,
                   GRD_SIZE, 1, 1, BLK_SIZE, 1, 1,
                    0, 0, args, 0);
    hipDeviceSynchronize();
}

int main(){
    // CuPtr<uint32_t> out_d(BLK_SIZE);
    // CuPtr<uint32_t> startClk_d(BLK_SIZE);
    // CuPtr<uint32_t> stopClk_d(BLK_SIZE);

    // for (int i = 0; i < WARMUP; i++) {
    //     smem_ld_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr());
    // }
    // hipDeviceSynchronize();

    // smem_ld_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr());
    // hipDeviceSynchronize();

    // HostPtr<uint32_t> out_h;
    // HostPtr<uint32_t> startClk_h;
    // HostPtr<uint32_t> stopClk_h;
    // startClk_d.ToHostPtr(startClk_h);
    // stopClk_d.ToHostPtr(stopClk_h);

    // double clockCycles_avg = 0.0;
    // for(int i = 0; i < BLK_SIZE; i++){
    //     std::cout << startClk_h(i) << " " << stopClk_h(i) << std::endl;
    //     clockCycles_avg += stopClk_h(i) - startClk_h(i);
    // }
    // clockCycles_avg /= BLK_SIZE;

    // printf("shared memory load latency: %lf cycles\n", clockCycles_avg / UNROLL);



    CuPtr<uint32_t> out_d(BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);

    // for (int i = 0; i < WARMUP; i++) {
    //     smem_st_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr());
    // }
    // hipDeviceSynchronize();

    smem_st_latency_test_run_drv(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr());
    // smem_st_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr());

    HostPtr<uint32_t> out_h;
    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);

    double clockCycles_avg = 0.0;
    for(int i = 0; i < BLK_SIZE; i++){
        std::cout << startClk_h(i) << " " << stopClk_h(i) << std::endl;
        clockCycles_avg += stopClk_h(i) - startClk_h(i);
    }
    clockCycles_avg /= BLK_SIZE;

    printf("shared memory store latency: %lf cycles\n", clockCycles_avg);
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/smem_latency_my smem_latency_my.cu

// nvcc --keep --keep-dir latency_midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/smem_latency_my smem_latency_my.cu
// cuasm --bin2asm latency_midRes/smem_latency_my.sm_86.cubin -o latency_midRes/smem_latency_my.sm_86.cuasm

// cuasm --asm2bin latency_midRes/smem_latency_my.sm_86.cuasm -o latency_midRes/smem_latency_my.sm_86.cubin





// ld结果是：22.9620 约等于 23 个时钟周期
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hostptr.hpp"

#include <iostream>
#include <cstdio>
#include <cstdint>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GRD_SIZE 1
#define BLK_SIZE 16
#define UNROLL 50
#define WARMUP 100

// SMem(ld/st): 23/19 Cycles
__global__ void smem_st_latency_test_kernel(uint32_t *startClk, uint32_t *stopClk, uint32_t *out){
    __shared__ uint32_t Arr[BLK_SIZE+UNROLL];
    int tid = threadIdx.x;
    uint32_t start, stop;
    uint32_t *arr_addr = Arr+tid;
    // 为了让计时区间的代码干净一些，先跑一次，准备好寄存器。
    // *arr_addr = tid; 也可以。主要还是改汇编。
    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        *(arr_addr+i) = tid;
    }

    asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");
    
    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        *(arr_addr+i) = tid;
    }

    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid] = stop;
    out[tid] = *(arr_addr+UNROLL-1);
}

void smem_st_latency_test_run_drv(uint32_t *startClk, uint32_t *stopClk, uint32_t *out){
    static hipModule_t cuModule;
    static hipFunction_t kernel;
    static bool isInitialized = false;

    if (!isInitialized)
    {
        hipInit(0);

        // Create module from binary file
        hipModuleLoad(&cuModule, "latency_midRes/smem_latency_st.sm_86.cubin");

        // Get function handle from module 
        hipModuleGetFunction(&kernel, cuModule, "_Z27smem_st_latency_test_kernelPjS_S_");

        isInitialized = true;
    }

    void* args[] = { (void*)&startClk, (void*)&stopClk, (void*)&out };
    hipModuleLaunchKernel(kernel,
                   GRD_SIZE, 1, 1, BLK_SIZE, 1, 1,
                    0, 0, args, 0);
    hipDeviceSynchronize();
}


void doStoreTest(){
    CuPtr<uint32_t> out_d(BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);
    // warm up
    for (int i = 0; i < WARMUP; i++) {
        smem_st_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr());
    }
    hipDeviceSynchronize();
    // run
    smem_st_latency_test_run_drv(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr());
    // smem_st_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr());

    HostPtr<uint32_t> out_h;
    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);

    double clockCycles = stopClk_h(0) - startClk_h(0);
    clockCycles /= UNROLL;

    printf("shared memory store latency: %lf cycles\n", clockCycles);
}

int main(){
    doStoreTest();
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/smem_latency_st smem_latency_st.cu


// nvcc --keep --keep-dir latency_midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/smem_latency_st smem_latency_st.cu
// cuasm --bin2asm latency_midRes/smem_latency_st.sm_86.cubin -o latency_midRes/smem_latency_st.sm_86.cuasm

// cuasm --asm2bin latency_midRes/smem_latency_st.sm_86.cuasm -o latency_midRes/smem_latency_st.sm_86.cubin

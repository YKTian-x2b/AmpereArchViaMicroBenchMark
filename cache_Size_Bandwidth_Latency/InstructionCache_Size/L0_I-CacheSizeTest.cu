#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>
using namespace std;

#define GRD_SIZE (30)
#define BLK_SIZE (128)
#define N_UNROLL (4)

__global__ void L0CacheSize_test_kernel(const float4 v, int n_unroll, unsigned *startClk, unsigned *stopClk, float *a)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int bdx = blockDim.x;
    int wid = tid / 32;
    int lid = tid % 32;

    float v0 = v.x;
    float v1 = v.y;
    float v2 = v.z;

    // 填充L0指令缓存 预热
    #pragma unroll
    for(int n = 0; n < n_unroll; n++)
        v0 = fmaf(v0, v2, v1);
    
    asm volatile ("bar.sync 0;");

    // 开始计时 start
    unsigned start = 0;
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");

    #pragma unroll
    for(int n = 0; n < n_unroll; n++)
        v0 = fmaf(v0, v2, v1);
    
    // 停止计时 stop
    unsigned stop = 0;
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    // 将时间和数据写回内存  
    // only first lane of warp writes to memory
    if(lid == 0){
        int index = ((bid * bdx) >> 5) + wid;
        a[index] = v0;
        startClk[index] = start;
        stopClk[index]= stop;
    }  
}

void L0CacheSize_test_run(const float4 v, int n_unroll, unsigned *startClk, unsigned *stopClk, float *a)
{
    L0CacheSize_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(v, n_unroll, startClk, stopClk, a);
    hipDeviceSynchronize();
}

void L0CacheSize_test_run_drv(const float4 v, int n_unroll, unsigned *startClk, unsigned *stopClk, float *a)
{
    static hipModule_t cuModule;
    static hipFunction_t kernel;
    static bool isInitialized = false;

    if (!isInitialized)
    {
        hipInit(0);

        // Create module from binary file
        hipModuleLoad(&cuModule, "L0CacheSizeTest.sm_86.cubin");

        // Get function handle from module _Z23L0CacheSize_test_kernel6float4iPjS0_Pf
        hipModuleGetFunction(&kernel, cuModule, "_Z23L0CacheSize_test_kernel6float4iPjS0_Pf");

        printf("cuModule = %#llx\n", (unsigned long long)cuModule);
        printf("cuFunction = %#llx\n", (unsigned long long)kernel);
        isInitialized = true;
    }


    void* args[] = { (void*)&v, (void*)&n_unroll, (void*)&startClk, (void*)&stopClk, (void*)&a };
    hipModuleLaunchKernel(kernel,
                   GRD_SIZE, 1, 1, BLK_SIZE, 1, 1,
                    0, 0, args, 0);
    hipDeviceSynchronize();
}

void doTest()
{
    size_t eleSize = GRD_SIZE * BLK_SIZE / 32;
    CuPtr<float> da(eleSize);
    CuPtr<unsigned> d_startClk(eleSize);
    CuPtr<unsigned> d_stopClk(eleSize);

    float4 v = make_float4(1.0f, 1.0f, 1.0f, 0);
    
    printf("### Runing...\n");

    da.SetZeros();
    d_startClk.SetZeros();
    d_stopClk.SetZeros();
    int n_unroll = N_UNROLL;

    L0CacheSize_test_run_drv(v, n_unroll, d_startClk.GetPtr(), d_stopClk.GetPtr(), da.GetPtr());
    

    printf("\n### Result checking...\n");
    HostPtr<float> ha;
    da.ToHostPtr(ha);
    HostPtr<unsigned> h_startClk;
    d_startClk.ToHostPtr(h_startClk);
    HostPtr<unsigned> h_stopClk;
    d_stopClk.ToHostPtr(h_stopClk);
    double avgCycles = 0;
    for(int i = 0; i < eleSize; i++)
    {
        unsigned elapsed = h_stopClk(i)-h_startClk(i);
        avgCycles += elapsed;
    }
    printf("avg Cycles: %lf\n", avgCycles/eleSize);
}

int main(){
    doTest();
    return 0;
}

// nvcc --keep -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I/opt/kaiProjects/GEMM_kai/Utils -L /usr/local/cuda/lib64 -l cuda -o L0CacheSizeTest save/L0CacheSizeTest.cu

// cuasm --bin2asm L0CacheSizeTest.sm_86.cubin

// cp L0CacheSizeTest.sm_86.cuasm save/L0CacheSizeTest.template.sm_86.cuasm && cp L0CacheSizeTest.sm_86.cuasm save/L0CacheSizeTest.origin.sm_86.cuasm

// @CUASM_INSERT_MARKER_POS.WORK_1

// python3 save/test_L0_Cache.py
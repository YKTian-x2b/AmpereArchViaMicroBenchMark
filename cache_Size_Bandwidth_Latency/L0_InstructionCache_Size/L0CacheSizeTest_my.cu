#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hostptr.hpp"
#include "cuptr.hpp"

#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GRD_SIZE 1
#define BLK_SIZE 128
#define N_UNROLL 10
#define WARM_UP 100

__global__ void L0CacheSize_test_kernel(const float4 v, unsigned *startClk, unsigned *stopClk, float *out){
    int tid = threadIdx.x;
    float v0 = v.x;
    float v1 = v.y;
    float v2 = v.z;
    // populate l0 i-cache
    #pragma unroll
    for(int n = 0; n < N_UNROLL; n++){
        v0 = fmaf(v0, v2, v1);
    }

    // 开始计时 start
    unsigned start, stop;
    asm volatile (
        "mov.u32 %0, %%clock;\n"
        : "=r"(start) : : "memory"
    );

    #pragma unroll
    for(int n = 0; n < N_UNROLL; n++){
        v0 = fmaf(v0, v2, v1);
    }
    
    // 停止计时 stop
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    // 将时间和数据写回内存  
    startClk[tid] = start;
    stopClk[tid]= stop;
    if(v0 == 0){
        out[tid] = v0;
    }  
}

float L0CacheSize_test_run(const float4 v, unsigned *startClk, unsigned *stopClk, float *out)
{
    hipEvent_t event_start, event_stop;
    checkCudaErrors(hipEventCreate(&event_start));
    checkCudaErrors(hipEventCreate(&event_stop));
    float elapsedTime;
    checkCudaErrors(hipEventRecord(event_start, 0));

    L0CacheSize_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(v, startClk, stopClk, out);

    checkCudaErrors(hipEventRecord(event_stop, 0));
    checkCudaErrors(hipEventSynchronize(event_stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, event_start, event_stop));
    checkCudaErrors(hipEventDestroy(event_start));
    checkCudaErrors(hipEventDestroy(event_stop));
    return elapsedTime;
}

float L0CacheSize_test_run_drv(const float4 v, unsigned *startClk, unsigned *stopClk, float *out)
{
    static hipModule_t cuModule;
    static hipFunction_t kernel;
    static bool isInitialized = false;

    if (!isInitialized)
    {
        hipInit(0);

        // Create module from binary file
        hipModuleLoad(&cuModule, "midRes_my/L0CacheSizeTest_my.sm_86.cubin");

        // Get function handle from module
        hipModuleGetFunction(&kernel, cuModule, "_Z23L0CacheSize_test_kernel6float4PjS0_Pf");

        isInitialized = true;
    }

    hipEvent_t event_start, event_stop;
    checkCudaErrors(hipEventCreate(&event_start));
    checkCudaErrors(hipEventCreate(&event_stop));
    float elapsedTime;
    checkCudaErrors(hipEventRecord(event_start, 0));

    void* args[] = { (void*)&v, (void*)&startClk, (void*)&stopClk, (void*)&out};
    hipModuleLaunchKernel(kernel,
                   GRD_SIZE, 1, 1, BLK_SIZE, 1, 1,
                    0, 0, args, 0);

    checkCudaErrors(hipEventRecord(event_stop, 0));
    checkCudaErrors(hipEventSynchronize(event_stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, event_start, event_stop));
    checkCudaErrors(hipEventDestroy(event_start));
    checkCudaErrors(hipEventDestroy(event_stop));
    return elapsedTime;
}

int main(){
    size_t eleSize = GRD_SIZE * BLK_SIZE;
    CuPtr<float> d_out(eleSize);
    CuPtr<unsigned> d_startClk(eleSize);
    CuPtr<unsigned> d_stopClk(eleSize);

    float4 v = make_float4(1.0f, 1.0f, 1.0f, 1.0f);

    printf("### Warming Up...\n");
    d_out.SetZeros();
    d_startClk.SetZeros();
    d_stopClk.SetZeros();
    // warm up
    // for(int i = 0; i < WARM_UP; i++){
    //     // L0CacheSize_test_run(v, d_startClk.GetPtr(), d_stopClk.GetPtr(), d_out.GetPtr());
    //     L0CacheSize_test_run_drv(v, d_startClk.GetPtr(), d_stopClk.GetPtr(), d_out.GetPtr());
    // }
    
    // L0CacheSize_test_run(v, d_startClk.GetPtr(), d_stopClk.GetPtr(), d_out.GetPtr());
    L0CacheSize_test_run_drv(v, d_startClk.GetPtr(), d_stopClk.GetPtr(), d_out.GetPtr());

    printf("### Runing...\n");
    d_out.SetZeros();
    d_startClk.SetZeros();
    d_stopClk.SetZeros();
    // float elapsedAll = L0CacheSize_test_run(v, d_startClk.GetPtr(), d_stopClk.GetPtr(), d_out.GetPtr());
    float elapsedAll = L0CacheSize_test_run_drv(v, d_startClk.GetPtr(), d_stopClk.GetPtr(), d_out.GetPtr());

    printf("\n### Result checking...\n");
    HostPtr<float> h_out;
    HostPtr<unsigned> h_startClk;
    HostPtr<unsigned> h_stopClk;
    d_out.ToHostPtr(h_out);
    d_startClk.ToHostPtr(h_startClk);
    d_stopClk.ToHostPtr(h_stopClk);
    double avgCycles = 0;
    for(int i = 0; i < eleSize; i++)
    {
        unsigned elapsed = h_stopClk(i)-h_startClk(i);
        avgCycles += elapsed;
    }
    printf("avg Cycles: %lf\n", avgCycles/eleSize);
}

// nvcc --keep --keep-dir midRes_my -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L0CacheSizeTest_my L0CacheSizeTest_my.cu

// cuasm --bin2asm midRes_my/L0CacheSizeTest_my.sm_86.cubin -o midRes_my/L0CacheSizeTest_my.sm_86.cuasm

// cp midRes_my/L0CacheSizeTest_my.sm_86.cuasm res/L0CacheSizeTest_my.template.sm_86.cuasm && cp midRes_my/L0CacheSizeTest_my.sm_86.cuasm res/L0CacheSizeTest_my.origin.sm_86.cuasm

// @CUASM_INSERT_MARKER_POS.

// python3 test_L0_Cache_my.py
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hostptr.hpp"
#include "cuptr.hpp"

#include <iostream>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define GRD_SIZE 1
#define BLK_SIZE 512
#define UNROLL 10

__global__ void L0CacheSize_test_kernel(){

}

int main(){

    // 预热L0/L1/L2 i-cache
    L0CacheSize_test_kernel<<<GRD_SIZE, BLK_SIZE>>>();
    hipDeviceSynchronize();

    // run
    L0CacheSize_test_kernel<<<GRD_SIZE, BLK_SIZE>>>();
    hipDeviceSynchronize();

}
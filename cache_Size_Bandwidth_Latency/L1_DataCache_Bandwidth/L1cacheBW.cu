#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include <iostream>
#include <random>
#include <ctime>
#include "hipblas.h"

// Bytes for total L1/SharedMem is 128*1024; L1 max is 128*1024B
#define L1_SIZE 96 * 256
#define THREADS_NUM 512
#define WARP_SIZE 32

float posArray[L1_SIZE];

__host__ void assignData(){
    std::default_random_engine e;
    e.seed(time(0));
    std::uniform_real_distribution<float> u(1, 65536);
    for(int i = 0; i < L1_SIZE; i++){
        posArray[i] = i;
    }
}

__global__ void l1_bw(uint32_t *startClk, uint32_t *stopClk, float *dsink, float *posArray){
    // 线程索引
    uint32_t tid = threadIdx.x;
    // 侧效变量，目的是避免编译器删除这段代码
    float sink = 0;
    // 通过填充 L1 缓存来预热 且 保证L1缓存了全部数据
    for (uint32_t i = tid; i < L1_SIZE; i += THREADS_NUM) {
        float * ptr = posArray+i;
        // sink += *ptr;
        asm volatile ("{\t\n"
        ".reg .f32 data;\n\t"
        "ld.global.ca.f32 data, [%1];\n\t"
        "add.f32 %0, data, %0;\n\t"
        "}" : "+f"(sink) : "l"(ptr): "memory"
        );
    }
    // 同步所有线程
    asm volatile ("bar.sync 0;");
    // 开始计时 uint32_t
    uint32_t start = 0;
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");
    // 从 L1 缓存加载数据，累加
    for (uint32_t i = 0; i < L1_SIZE; i += THREADS_NUM) {
        float * ptr = posArray+i;
        // 每个warp都会加载一级缓存中的所有数据
        for (uint32_t j = 0; j < THREADS_NUM; j += WARP_SIZE) {
            uint32_t offset = (tid+j)%THREADS_NUM;
            // sink += ptr[offset];
            asm volatile ("{\t\n"
                ".reg .f32 data;\n\t" 
                "ld.global.ca.f32 data, [%1];\n\t"
                "add.f32 %0, data, %0;\n\t"
                "}" : "+f"(sink) : "l"(ptr+offset) : "memory"
            );
        }
    }
    // 同步所有线程
    asm volatile ("bar.sync 0;");
    // 停止计时 uint32_t
    uint32_t stop = 0;
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");
    // 将时间和数据写回内存
    startClk[tid] = start;
    stopClk[tid]= stop;
    dsink[tid]= sink;
}

int main(){
    assignData();

    unsigned arrBytes = sizeof(float) * L1_SIZE;
    unsigned clkBytes = sizeof(uint32_t) * THREADS_NUM;
    unsigned sinkBytes = sizeof(float) * THREADS_NUM;

    uint32_t *d_startClk, *d_stopClk;
    float *d_arr, *dsink;
    uint32_t *h_startClk, *h_stopClk;
    float *hsink;

    checkCudaErrors(hipMalloc(&d_arr, arrBytes));
    checkCudaErrors(hipMalloc(&d_startClk, clkBytes));
    checkCudaErrors(hipMalloc(&d_stopClk, clkBytes));
    checkCudaErrors(hipMalloc(&dsink, sinkBytes));
    checkCudaErrors(hipMemcpy(d_arr, posArray, arrBytes, hipMemcpyHostToDevice));

    l1_bw<<<1, THREADS_NUM>>>(d_startClk, d_stopClk, dsink, d_arr);
    hipDeviceSynchronize();
   
    h_startClk = (uint32_t*)malloc(clkBytes);
    h_stopClk = (uint32_t*)malloc(clkBytes);
    hsink = (float*)malloc(sinkBytes);
    checkCudaErrors(hipMemcpy(h_startClk, d_startClk, clkBytes, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_stopClk, d_stopClk, clkBytes, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(hsink, dsink, sinkBytes, hipMemcpyDeviceToHost));

    float clockCycles_avg = 0.0;
    float sink_total = 0.0;
    for(int i = 0; i < THREADS_NUM; i++){
        clockCycles_avg += h_stopClk[i] - h_startClk[i];
        // std::cout << h_startClk[i] << " " << h_stopClk[i] << std::endl;
        sink_total += hsink[i];
    }
    clockCycles_avg /= THREADS_NUM;
    std::cout << h_startClk[0] << " " << h_stopClk[0] << std::endl;
    std::cout << "clockCycles_avg: " << clockCycles_avg << std::endl; // 63433
    std::cout << "sink_total: " << sink_total << std::endl;
    // 每个warp都会加载一级缓存中的所有数据 一个SM启动了 128/32个warp 所以一共读取了BytesAll个字节 per SM
    uint32_t BytesAll = L1_SIZE * sizeof(float) * (THREADS_NUM / WARP_SIZE); 
    // 理论上限是64B 16个LD_ST单元 * 4B/LD_ST单元/周期 实测结果是62B左右
    float BpCpSM = BytesAll / clockCycles_avg;
    std::cout << "L1 real throughput is  " << BpCpSM <<  " Bytes per cycle per SM" << std::endl;

    checkCudaErrors(hipFree(d_arr));
    checkCudaErrors(hipFree(d_startClk));
    checkCudaErrors(hipFree(d_stopClk));
    checkCudaErrors(hipFree(dsink));
    free(h_startClk);
    free(h_stopClk);
    free(hsink);
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L1cacheBW L1cacheBW.cu
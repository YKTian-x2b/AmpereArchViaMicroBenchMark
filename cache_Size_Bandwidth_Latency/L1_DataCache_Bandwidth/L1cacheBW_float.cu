#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hostptr.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdio>

#define WARP_SIZE 32
// 得保证数据能被L1_D-Cache装下
// Bytes for total L1/SharedMem is 128*1024B; L1 max is 128*1024B；这里用96*1024B；
#define N_DATA 96 * 256
#define BLK_SIZE 1024
#define GRD_SIZE 1

__host__ void assignData(float *posArray){
    for(int i = 0; i < N_DATA+BLK_SIZE; i++){
        posArray[i] = 1;
    }
}

__device__ __forceinline__
float ldg_ca(const void *ptr) {
    float ret;
    asm volatile (
        "ld.global.ca.f32 %0, [%1];"
        : "=f"(ret)
        : "l"(ptr)
    );
    return ret;
}

__global__ void L1DatacacheBW_test_kernel(uint32_t *startClk, uint32_t *stopClk, float *dsink, float *Arr){
    int tid = threadIdx.x;
    const float* ldg_ptr = Arr + tid;

    float sink = 0;
    // 保证L1缓存了全部数据
    for (int i = 0; i < N_DATA; i += WARP_SIZE) {
        sink += ldg_ca(ldg_ptr + i);
    }
    asm volatile ("bar.sync 0;");

    sink = 0;
    uint32_t start, stop;
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");
    // LDG指令发射有4个cycle的间隔且 FADD和LDG不是一个port 所以TLP+ILP情况下 基本不影响满流水的形成
    for (int i = 0; i < N_DATA; i += WARP_SIZE) {
        sink += ldg_ca(ldg_ptr + i);
    }
    // 因为要小心准确的获取整个SM的吞吐，所以这里应该有一个同步，保证最后测得的时间间隔是整个Block的开始-结束时间，不是某个warp的，也不是所有warp的均值。
    // asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid]= stop;
    dsink[tid]= sink;
}

int main(){
    // 为了让kernel的sass中ldg指令占比尽量高，我们把kernel写的干净一点，尾数据(+BLK_SIZE)最多可以保证每个thread读取全部数据而不出现问题
    HostPtr<float> arr_h(N_DATA+BLK_SIZE);
    assignData(arr_h.GetPtr());
    CuPtr<float> arr_d(arr_h);
    CuPtr<float> sink_d(BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);
    // warm up
    L1DatacacheBW_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), sink_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();

    // run
    L1DatacacheBW_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), sink_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();
   
    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    HostPtr<float> sink_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);
    sink_d.ToHostPtr(sink_h);

    uint32_t minStartClk = ~0x0;
    uint32_t maxStopClk = 0;
    float sink_total = 0.0;
    // 需要注意的是 每cycle字节数 需要cycle尽量精准。
    // 如果在kernel访存结束的时候没有同步Block，且这里用了均值时间，就会发现实测值超过理论值，就是因为用的时间不够精准。
    // 最佳方式应该是 在kernel访存结束的时候没有同步Block。
    // 这里选择的方式是不同步，但用最大时间间隔。
    for(int i = 0; i < BLK_SIZE; i++){
        sink_total += sink_h(i);
        if(startClk_h(i) < minStartClk){
            minStartClk = startClk_h(i);
        }
        if(stopClk_h(i) > maxStopClk){
            maxStopClk = stopClk_h(i);
        }
        // if(i % 32 == 0)
        //     std::cout << i << ": " << startClk_h(i) << " " << stopClk_h(i) << std::endl;
    }
    float clockCycles = maxStopClk - minStartClk;

    std::cout << "clockCycles: " << clockCycles << std::endl;
    std::cout << "sink_total: " << sink_total << std::endl;
    // warp加载了全部的L1 cache Data
    uint32_t BytesAll = N_DATA * sizeof(float) * (BLK_SIZE/WARP_SIZE);  // * (BLK_SIZE/WARP_SIZE);   //  * BLK_SIZE;
    float BpCpSM = BytesAll / clockCycles;
    std::cout << "L1 Data cache Bandwidth is  " << BpCpSM <<  " Bytes per cycle per SM" << std::endl;
    std::cout << "L1 Data cache Bandwidth is  " << BpCpSM * 1.7 <<  " GBps per SM" << std::endl;
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L1cacheBW_float L1cacheBW_float.cu

// nvcc --keep --keep-dir midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L1cacheBW_float L1cacheBW_float.cu
// cuasm --bin2asm midRes/L1cacheBW_float.sm_86.cubin -o midRes/L1cacheBW_float.sm_86.cuasm

// L1 Data cache Bandwidth is  63.5629 Bytes per cycle per SM
// L1 Data cache Bandwidth is  108.057 GBps per SM
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hostptr.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdio>

#define WARP_SIZE 32
// 得保证数据能被L1_D-Cache装下
// Bytes for total L1/SharedMem is 128*1024B; L1 max is 128*1024B；这里用64*1024B；
#define N_DATA 32 * 256
#define BLK_SIZE 1024
#define GRD_SIZE 1

__host__ void assignData(float2 *posArray){
    for(int i = 0; i < N_DATA+BLK_SIZE; i++){
        posArray[i] = make_float2(i+1.0, i+2.0);
    }
}

__device__ __forceinline__
float2 ldg_ca_v2(const void *ptr) {
    float2 ret;
    asm volatile (
        "ld.global.ca.v2.f32 {%0, %1}, [%2];"
        : "=f"(ret.x), "=f"(ret.y)
        : "l"(ptr)
    );
    return ret;
}

__global__ void L1DatacacheBW_test_kernel(uint32_t *startClk, uint32_t *stopClk, float2 *dsink, float2 *Arr){
    int tid = threadIdx.x;
    const float2* ldg_ptr = Arr + tid;

    float2 sink = make_float2(0.0, 0.0);
    // 保证L1缓存了全部数据
    for (int i = 0; i < N_DATA; i += BLK_SIZE) {
        float2 res = ldg_ca_v2(ldg_ptr + i);
        sink.y += res.x + res.y;
    }

    uint32_t start, stop;
    asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");
    // LDG指令发射有4个cycle的间隔且 FADD和LDG不是一个port 所以TLP+ILP情况下 基本不影响满流水的形成
    #pragma unroll
    for (int i = 0; i < N_DATA; i += WARP_SIZE) {
        float2 res = ldg_ca_v2(ldg_ptr + i);
        sink.x += res.x + res.y;
    }
    // 因为要小心准确的获取整个SM的吞吐，所以这里应该有一个同步，保证最后测得的时间间隔是整个Block的开始-结束时间，不是某个warp的，也不是所有warp的均值。
    // asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid]= stop;
    dsink[tid]= sink;
}

int main(){
    // 为了让kernel的sass中ldg指令占比尽量高，我们把kernel写的干净一点，尾数据(+BLK_SIZE)最多可以保证每个thread读取全部数据而不出现问题
    HostPtr<float2> arr_h(N_DATA+BLK_SIZE);
    assignData(arr_h.GetPtr());
    CuPtr<float2> arr_d(arr_h);
    CuPtr<float2> sink_d(BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);
    // warm up
    L1DatacacheBW_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), sink_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();

    // run
    L1DatacacheBW_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), sink_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();
   
    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    HostPtr<float2> sink_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);
    sink_d.ToHostPtr(sink_h);

    uint32_t minStartClk = ~0x0;
    uint32_t maxStopClk = 0;
    float sink_total = 0.0;
    // 需要注意的是 每cycle字节数 需要cycle尽量精准。
    // 如果在kernel访存结束的时候没有同步Block，且这里用了均值时间，就会发现实测值超过理论值，就是因为用的时间不够精准。
    // 最佳方式应该是 在kernel访存结束的时候没有同步Block。
    // 这里选择的方式是不同步，但用最大时间间隔。
    for(int i = 0; i < BLK_SIZE; i++){
        sink_total += sink_h(i).x;
        if(startClk_h(i) < minStartClk){
            minStartClk = startClk_h(i);
        }
        if(stopClk_h(i) > maxStopClk){
            maxStopClk = stopClk_h(i);
        }
        // if(i % 32 == 0)
        //     std::cout << i << ": " << startClk_h(i) << " " << stopClk_h(i) << std::endl;
    }
    float clockCycles = maxStopClk - minStartClk;

    std::cout << "clockCycles: " << clockCycles << std::endl;
    std::cout << "sink_total: " << sink_total << std::endl;
    // warp加载了全部的L1 cache Data
    uint32_t BytesAll = N_DATA * sizeof(float2) * (BLK_SIZE/WARP_SIZE);  // * (BLK_SIZE/WARP_SIZE);   //  * BLK_SIZE;
    float BpCpSM = BytesAll / clockCycles;
    std::cout << "L1 Data cache Bandwidth is  " << BpCpSM <<  " Bytes per cycle per SM" << std::endl;
    std::cout << "L1 Data cache Bandwidth is  " << BpCpSM * 1.7 <<  " GBps per SM" << std::endl;
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L1cacheBW_float2 L1cacheBW_float2.cu

// nvcc --keep --keep-dir midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L1cacheBW_float2 L1cacheBW_float2.cu
// cuasm --bin2asm midRes/L1cacheBW_float2.sm_86.cubin -o midRes/L1cacheBW_float2.sm_86.cuasm

// L1 Data cache Bandwidth is  124.652 Bytes per cycle per SM
// L1 Data cache Bandwidth is  211.909 GBps per SM
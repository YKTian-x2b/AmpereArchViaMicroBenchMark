#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hostptr.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdio>

#define WARP_SIZE 32
// Bytes for total L1/SharedMem is 128*1024; L1 max is 128*1024B
#define N_DATA 96 * 256
#define BLK_SIZE 1024
#define GRD_SIZE 1


__host__ void assignData(float *posArray){
    for(int i = 0; i < N_DATA; i++){
        posArray[i] = 1;
    }
}

__device__ __forceinline__
float ldg_ca(const void *ptr) {
    float ret;
    asm volatile (
        "ld.global.ca.b32 %0, [%1];"
        : "=f"(ret)
        : "l"(ptr)
    );
    return ret;
}

__global__ void L1DatacacheBW_test_kernel(uint32_t *startClk, uint32_t *stopClk, float *dsink, float *Arr){
    int tid = threadIdx.x;
    const float *ldg_ptr = Arr + tid;

    float sink = 0;
    // 通过填充 L1 缓存来预热 且 保证L1缓存了全部数据
    for (int i = 0; i < N_DATA; i += WARP_SIZE) {
        sink += ldg_ca(ldg_ptr + i);
    }
    asm volatile ("bar.sync 0;");

    sink = 0;
    uint32_t start = 0;
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");

    for (int i = 0; i < N_DATA; i += WARP_SIZE) {
        sink += ldg_ca(ldg_ptr + i);
    }

    uint32_t stop = 0;
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid]= stop;
    dsink[tid]= sink;
}

int main(){
    HostPtr<float> arr_h(N_DATA);
    assignData(arr_h.GetPtr());
    CuPtr<float> arr_d(arr_h);
    CuPtr<float> sink_d(BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);

    L1DatacacheBW_test_kernel<<<1, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), sink_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();
   
    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    HostPtr<float> sink_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);
    sink_d.ToHostPtr(sink_h);

    float clockCycles_avg = 0.0;
    float sink_total = 0.0;
    for(int i = 0; i < BLK_SIZE; i++){
        clockCycles_avg += stopClk_h(i) - startClk_h(i);
        // std::cout << h_startClk[i] << " " << h_stopClk[i] << std::endl;
        sink_total += sink_h(i);
    }
    clockCycles_avg /= BLK_SIZE;

    std::cout << "clockCycles_avg: " << clockCycles_avg << std::endl;
    std::cout << "sink_total: " << sink_total << std::endl;
    // Block加载了全部的L1 cache Data
    uint32_t BytesAll = N_DATA * sizeof(float) * (BLK_SIZE/WARP_SIZE);  //  
    // 理论上限是64B 16个LD_ST单元 * 4B/LD_ST单元/周期 实测结果是 65B左右
    float BpCpSM = BytesAll / clockCycles_avg;
    std::cout << "L1 Data cache Bandwidth is  " << BpCpSM <<  " Bytes per cycle per SM" << std::endl;
    std::cout << "L1 Data cache Bandwidth is  " << BpCpSM * 1.7 <<  " GBps per SM" << std::endl;
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L1cacheBW_myDefine L1cacheBW_myDefine.cu

// nvcc --keep --keep-dir midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L1cacheBW_myDefine L1cacheBW_myDefine.cu
// cuasm --bin2asm midRes/L1cacheBW_myDefine.sm_86.cubin -o midRes/L1cacheBW_myDefine.sm_86.cuasm


// 512: L1 Data cache Bandwidth is  63.7894 Bytes per cycle per SM
// 1024: L1 Data cache Bandwidth is  65.3869 Bytes per cycle per SM
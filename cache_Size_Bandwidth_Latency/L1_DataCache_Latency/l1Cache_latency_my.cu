#include "hip/hip_runtime.h"
#include "cuptr.hpp"
#include "hostptr.hpp"
#include "hip/hip_runtime_api.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

#define GRD_SIZE 1
// 16*8=128字节
// 这里的BLK_SIZE最大是16，取32就会使测试结果从33cycles变为35cycles。可能是因为两次L1事务中间需要间隔2个cycle吗？
// 结合L1的吞吐峰值是一个cycle64B，那么发射指令之后的两个cycle就会读两次64B作为一个整体结束，然后再读两次64B作为一个整体结束吗？
// 所以结束的时间点是按 每两次读取完全结束的时间算的，这两次就叫 一个transaction 吗？
#define BLK_SIZE 16
// [10, 50, 100, 200, 400] 别超过L0_I-Cache 都可以试试 结果差不多
#define UNROLL 10
#define WARM_UP 100

__global__ void l1Cache_latency_test_kernel(uint32_t *startClk, uint32_t *stopClk, void **out, void **arr){
    int tid = threadIdx.x;
    void **ldg_ptr = arr + tid;

    // fill in l1 cache
    for(int i = 0; i < UNROLL; i++){
        asm volatile (
            "ld.global.ca.nc.b64 %0, [%0];\n"
            : "+l"(ldg_ptr)
            : : "memory"
        );
    }

    uint32_t start, stop;
    asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");

    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        asm volatile (
            "ld.global.ca.nc.b64 %0, [%0];\n"
            : "+l"(ldg_ptr)
            : : "memory"
        );
    }
    
    // 不用同步的原因有三：只有一个warp；保持kernel的sass干净；latency只需测本warp的时间间隔，不需要整个SM/Block的性能；
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid] = stop;
    if(ldg_ptr == nullptr){
        *out = ldg_ptr;
    }
}

int main(){
    // 为了非常精准的测试latency，理想情况需要，两个计时clock之间，全都是背靠背的访存指令
    // 所以这里除了要用pointerChase，还要让读取结果直接就是指针，而不是偏移量，移除掉指针计算的那条指令
    // 太精彩了写的
    CuPtr<void*> arr_d(BLK_SIZE);
    HostPtr<void*> arr_h(BLK_SIZE);
    void **h_ptr = arr_h.GetPtr();
    void **d_ptr = arr_d.GetPtr();
    for(int i = 0; i < BLK_SIZE; i++){
        h_ptr[i] = (void*)(d_ptr + i);
    }
    uint32_t arrBytes = BLK_SIZE * sizeof(void*);
    checkCudaErrors(hipMemcpy(d_ptr, h_ptr, arrBytes, hipMemcpyHostToDevice));

    CuPtr<void*> out_d(BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);

    // warmup i-cache
    for(int i = 0; i < WARM_UP; i++){
        l1Cache_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    }
    hipDeviceSynchronize();

    // 
    l1Cache_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();

    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);

    double clockCycles = stopClk_h(0) - startClk_h(0);
    clockCycles /= UNROLL;
    std::cout << "l1 cache access latency: " << clockCycles << " cycles" << std::endl;
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/l1Cache_latency_my l1Cache_latency_my.cu

// nvcc --keep --keep-dir midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/l1Cache_latency_my l1Cache_latency_my.cu
// cuasm --bin2asm midRes/l1Cache_latency_my.sm_86.cubin midRes/l1Cache_latency_my.sm_86.cuasm

// l1 cache access latency: 33 cycles
#include "hip/hip_runtime.h"
#include "cuptr.hpp"
#include "hostptr.hpp"
#include "hip/hip_runtime_api.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

#define GRD_SIZE 1
#define BLK_SIZE 4
#define UNROLL 50
#define WARM_UP 100

__global__ void l1Cache_latency_test_kernel(uint32_t *startClk, uint32_t *stopClk, void **out, void **arr){
    int tid = threadIdx.x;
    void ** ldg_ptr = arr + tid;

    // fill in l1 cache
    for(int i = 0; i < UNROLL; i++){
        // asm volatile (
        //     "ld.global.ca.b64 %0, [%0];\n"
        //     : "+l"(ldg_ptr)
        //     : : "memory"
        // );
        asm volatile (
            "ld.global.nc.b64 %0, [%0];\n"
            : "+l"(ldg_ptr)
            : : "memory"
        );
    }

    uint32_t start, stop;
    asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");

    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        // asm volatile (
        //     "ld.global.ca.b64 %0, [%0];\n"
        //     : "+l"(ldg_ptr)
        //     : : "memory"
        // );
        asm volatile (
            "ld.global.nc.b64 %0, [%0];\n"
            : "+l"(ldg_ptr)
            : : "memory"
        );
    }
    

    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid] = stop;

    if(ldg_ptr == nullptr){
        *out = ldg_ptr;
    }
}

int main(){
    CuPtr<void*> arr_d(BLK_SIZE);
    HostPtr<void*> arr_h(BLK_SIZE);
    void **h_ptr = arr_h.GetPtr();
    void **d_ptr = arr_d.GetPtr();
    for(int i = 0; i < BLK_SIZE; i++){
        h_ptr[i] = (void*)(d_ptr + i);
    }
    uint32_t arrBytes = BLK_SIZE * sizeof(void*);
    checkCudaErrors(hipMemcpy(d_ptr, h_ptr, arrBytes, hipMemcpyHostToDevice));

    CuPtr<void*> out_d(BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);

    // warmup i-cache
    for(int i = 0; i < WARM_UP; i++){
        l1Cache_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    }
    hipDeviceSynchronize();

    // 
    l1Cache_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();

    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);

    double clockCycles_avg = 0.0;
    for(int i = 0; i < BLK_SIZE; i++){
        // std::cout << startClk_h(i) << " " << stopClk_h(i) << std::endl;
        clockCycles_avg += stopClk_h(i) - startClk_h(i);
    }
    clockCycles_avg /= (BLK_SIZE * UNROLL);
    std::cout << "l1 cache access latency: " << clockCycles_avg << " cycles" << std::endl;
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/l1Cache_latency_my l1Cache_latency_my.cu

// nvcc --keep --keep-dir midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/l1Cache_latency_my l1Cache_latency_my.cu
// cuasm --bin2asm midRes/l1Cache_latency_my.sm_86.cubin midRes/l1Cache_latency_my.sm_86.cuasm

// l1 cache access latency: 33 cycles
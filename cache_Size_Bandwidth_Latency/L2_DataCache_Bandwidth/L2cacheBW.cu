#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hostptr.hpp"

#include <iostream>
#include <random>
#include <ctime>

#include "hipblas.h"

// Bytes for L2_SIZE is 3 * 1024 * 1024 B
#define L2_SIZE 1024 * 256
#define GRD_SIZE 1800
#define THREADS_NUM 512
#define WARP_SIZE 32

float posArray[L2_SIZE];

__host__ void assignData(){
    std::default_random_engine e;
    e.seed(time(0));
    std::uniform_real_distribution<float> u(1, 65536);
    for(int i = 0; i < L2_SIZE; i++){
        posArray[i] = i;
    }
}

__global__ void l2_bw(uint32_t *startClk, uint32_t *stopClk, float *dsink, float *posArray){
    // 块和线程索引 
    uint32_t tid = threadIdx.x;
    uint32_t bid = blockIdx.x;
    //
    float sink = 0;
    // 通过填充 L2 缓存来预热 且保证L2缓存了全部数据
    for (uint32_t i = tid; i < L2_SIZE; i += THREADS_NUM) {
        float * ptr = posArray+i;
        // sink += *ptr;
        asm volatile ("{\t\n"
            ".reg .f32 data;\n\t"
            "ld.global.cg.f32 data, [%1];\n\t"
            "add.f32 %0, data, %0;\n\t"
            "}" : "+f"(sink) : "l"(ptr): "memory"
        );
    }
    // 同步所有线程
    asm volatile ("bar.sync 0;");
    // 开始计时 uint32_t
    uint32_t start = 0;
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");
    // 从二级缓存中加载数据并累加
    for (uint32_t i = 0; i < L2_SIZE; i += THREADS_NUM) {
        float* ptr = posArray+i;
        // 每次warp都会加载二级缓存中的所有数据
        for (uint32_t j = 0; j < THREADS_NUM; j += WARP_SIZE){
            uint32_t offset = (tid+j)%THREADS_NUM;
            asm volatile ("{\t\n"
                ".reg .f32 data;\n\t" 
                "ld.global.cg.f32 data, [%1];\n\t"
                "add.f32 %0, data, %0;\n\t"
                "}" : "+f"(sink) :"l"(ptr+offset) :"memory"
            );
        }
    }
    // 同步所有线程
    asm volatile ("bar.sync 0;");
    // 停止计时 uint32_t
    uint32_t stop = 0;
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");
    // 将时间和数据写回内存
    startClk[tid] = start;
    stopClk[tid]= stop;
    dsink[tid] = sink;
}

int main(){
    assignData();

    CuPtr<float> d_arr(L2_SIZE);
    CuPtr<uint32_t> d_startClk(THREADS_NUM);
    CuPtr<uint32_t> d_stopClk(THREADS_NUM);
    CuPtr<float> dsink(THREADS_NUM);
    
    unsigned arrBytes = sizeof(float) * L2_SIZE;
    checkCudaErrors(hipMemcpy(d_arr.GetPtr(), posArray, arrBytes, hipMemcpyHostToDevice));

    l2_bw<<<GRD_SIZE, THREADS_NUM>>>(d_startClk.GetPtr(), d_stopClk.GetPtr(), dsink.GetPtr(), d_arr.GetPtr());
    hipDeviceSynchronize();
   
    HostPtr<uint32_t> h_startClk;
    HostPtr<uint32_t> h_stopClk;
    HostPtr<float> hsink;
    d_startClk.ToHostPtr(h_startClk);
    d_stopClk.ToHostPtr(h_stopClk);
    dsink.ToHostPtr(hsink);

    float clockCycles_avg = 0.0;
    float sink_total = 0.0;
    for(int i = 0; i < THREADS_NUM; i++){
        clockCycles_avg += h_stopClk(i) - h_startClk(i);
        // std::cout << h_startClk(i) << " " << h_stopClk(i) << std::endl;
        sink_total += hsink(i);
    }
    clockCycles_avg /= THREADS_NUM;
    std::cout << h_startClk(0) << " " << h_stopClk(0) << std::endl;
    std::cout << "clockCycles_avg: " << clockCycles_avg << std::endl; 
    std::cout << "sink_total: " << sink_total << std::endl;
    // 每个warp都会加载一级缓存中的所有数据 一个SM启动了 128/32个warp 所以一共读取了BytesAll个字节 per SM
    uint32_t BytesAll = L2_SIZE * sizeof(float) * (THREADS_NUM / WARP_SIZE); 
    // 理论上限是64B 16个LD_ST单元 * 4B/LD_ST单元/周期 实测结果是62B左右
    float BpCpSM = BytesAll / clockCycles_avg;
    std::cout << "L2 real throughput is  " << BpCpSM <<  " Bytes per cycle per SM" << std::endl;
    float Bandwidth = BpCpSM * (THREADS_NUM / WARP_SIZE)
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L2cacheBW L2cacheBW.cu
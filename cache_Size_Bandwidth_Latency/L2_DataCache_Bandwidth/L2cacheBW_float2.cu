#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hostptr.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdio>

#define WARMUP_ITER 200
#define BENCH_ITER 200
#define WARP_SIZE 32
#define BLK_SIZE 512
// 希望是90（SM个数 * 一个SM能启动的ThreadBlock数） 的倍数，且要超大才能形成有效流水线
#define GRD_SIZE 90000
#define UNROLL 16
// L2 cache size 3*1024*1024字节 3*1024*256个元素
const size_t N_DATA = 1024*256;

__device__ __forceinline__
float2 ldg_cg_v2(const void *ptr) {
    float2 ret;
    asm volatile (
        "ld.global.cg.v2.f32 {%0, %1}, [%2];"
        : "=f"(ret.x), "=f"(ret.y)
        : "l"(ptr)
    );
    return ret;
}

__global__ void L2cacheBW_float2_kernel(const float2 *Arr, float *out) {
    int offset = (BLK_SIZE * UNROLL * blockIdx.x + threadIdx.x) % N_DATA;
    const float2 *ldg_ptr = Arr + offset;
    float sum = 0;

    #pragma unroll
    for (int i = 0; i < UNROLL; ++i) {
        float2 res = ldg_cg_v2(ldg_ptr + BLK_SIZE * i);
        sum += res.x + res.y;
    }

    if(sum != 0){
        *out = sum;
    }
}

int main(){
    // 添加尾数据，保证核函数正确且干净。
    HostPtr<float2> arr_h(N_DATA + BLK_SIZE * UNROLL);
    arr_h.SetZeros();
    CuPtr<float2> arr_d(arr_h);
    CuPtr<float> out_d(BLK_SIZE);

    std::cout << "Warming Up ..." << std::endl;
    // warm up to cache data into L2
    for (int i = 0; i < WARMUP_ITER; ++i) {
        L2cacheBW_float2_kernel<<<GRD_SIZE, BLK_SIZE>>>(arr_d.GetPtr(), out_d.GetPtr());
    }

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float time_ms;
    std::cout << "Running ..." << std::endl;
    checkCudaErrors(hipEventRecord(start));
    for (int i = 0; i < BENCH_ITER ; ++i) {
        L2cacheBW_float2_kernel<<<GRD_SIZE, BLK_SIZE>>>(arr_d.GetPtr(), out_d.GetPtr());
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&time_ms, start, stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    double gbps = ((double)((GRD_SIZE*BLK_SIZE*UNROLL) * sizeof(float2)) / 1e9) /
                  ((double)time_ms / BENCH_ITER / 1e3);
    printf("L2 cache bandwidth: %lfGB/s\n", gbps);
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L2cacheBW_float2 L2cacheBW_float2.cu

// 驱动调试
// nvcc --keep --keep-dir midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L2cacheBW_float2 L2cacheBW_float2.cu
// cuasm --bin2asm midRes/L2cacheBW_float2.sm_86.cubin -o midRes/L2cacheBW_float2.sm_86.cuasm


// GRD_SIZE90000的结果
// L2 cache bandwidth: 1312.528804GB/s
// GRD_SIZE9000的结果
// L2 cache bandwidth: 1266.337617GB/s

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hostptr.hpp"

#include <iostream>
#include <random>
#include <ctime>

#include "hipblas.h"

#define GRD_SIZE 900
#define BLK_SIZE 512
#define WARP_SIZE 32
// MAX Bytes for L2_SIZE is 3 * 1024 * 1024 B
#define L2_SIZE (GRD_SIZE * BLK_SIZE)

float posArray[L2_SIZE];

__host__ void assignData(){
    std::default_random_engine e;
    e.seed(time(0));
    std::uniform_real_distribution<float> u(1, 65536);
    for(int i = 0; i < L2_SIZE; i++){
        posArray[i] = i;
    }
}

// 与访存延迟相比，浮点加消耗的时间可以忽略不计。
__global__ void l2_bw(float *dsink, float *posArray){
    // 块和线程索引 
    uint32_t tid = threadIdx.x;
    uint32_t bid = blockIdx.x;
    //
    float sink = 0;

    // 从二级缓存中加载数据并累加
    for (uint32_t i = 0; i < L2_SIZE; i += BLK_SIZE) {
        float* ptr = posArray+i;
        // 每次warp都会加载二级缓存中的所有数据
        for (uint32_t j = 0; j < BLK_SIZE; j += WARP_SIZE){
            uint32_t offset = (tid+j)%BLK_SIZE;
            asm volatile ("{\t\n"
                ".reg .f32 data;\n\t" 
                "ld.global.cg.f32 data, [%1];\n\t"
                "add.f32 %0, data, %0;\n\t"
                "}" : "+f"(sink) :"l"(ptr+offset) :"memory"
            );
        }
    }
    // 同步所有线程
    asm volatile ("bar.sync 0;");
    dsink[tid] = sink;
}

int main(){
    assignData();

    CuPtr<float> d_arr(L2_SIZE);
    CuPtr<float> dsink(BLK_SIZE);
    
    unsigned arrBytes = sizeof(float) * L2_SIZE;
    checkCudaErrors(hipMemcpy(d_arr.GetPtr(), posArray, arrBytes, hipMemcpyHostToDevice));

    std::cout << "WarmUp ing ..." << std::endl;
    l2_bw<<<GRD_SIZE, BLK_SIZE>>>(dsink.GetPtr(), d_arr.GetPtr());
    hipDeviceSynchronize();

    hipEvent_t startE, stopE;
    float elapsedTime;
    checkCudaErrors(hipEventCreate(&startE));
    checkCudaErrors(hipEventCreate(&stopE));
    std::cout << "Running ..." << std::endl;
    checkCudaErrors(hipEventRecord(startE, 0));

    l2_bw<<<GRD_SIZE, BLK_SIZE>>>(dsink.GetPtr(), d_arr.GetPtr());

    checkCudaErrors(hipEventRecord(stopE, 0));
    checkCudaErrors(cudaEvnetSynchronize(stopE));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, event_start, event_stop));
    checkCudaErrors(hipEventDestroy(event_start));
    checkCudaErrors(hipEventDestroy(event_stop));

  
    HostPtr<float> hsink;
    dsink.ToHostPtr(hsink);
    float sink_total = 0.0;
    for(int i = 0; i < BLK_SIZE; i++){
        sink_total += hsink(i);
    }
    std::cout << "sink_total: " << sink_total << std::endl;

    // 每个warp都会加载一级缓存中的所有数据
    unsigned long long  BytesAll = L2_SIZE * sizeof(float) * (GRD_SIZE * BLK_SIZE / WARP_SIZE); 
    // 理论上限 16个LD_ST单元 * 30个SM * 4B/LD_ST单元/周期 * 1702M = 3.26784TB/s
    float Bandwidth = (BytesAll / elapsedTime) * 1000;
    std::cout << "L2 data cache Bandwidth is: " << Bandwidth << "B/s" << std::endl;
    
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L2cacheBW L2cacheBW.cu
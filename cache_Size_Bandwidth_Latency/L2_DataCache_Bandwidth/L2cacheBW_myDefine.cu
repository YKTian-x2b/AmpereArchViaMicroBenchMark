#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hostptr.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdio>

#define WARMUP_ITER 200
#define BENCH_ITER 200
#define WARP_SIZE 32
#define BLK_SIZE 512
// 希望是 （SM个数 * 一个SM能启动的ThreadBlock数） 90 的倍数 要超大才能形成有效流水线
#define GRD_SIZE 90000
#define UNROLL 16
// L2 cache size 3*1024*1024字节 3*1024*256个元素
const size_t N_DATA = 2*1024*256;

__device__ __forceinline__
float ldg_cg(const void *ptr) {
    float ret;
    asm volatile (
        "ld.global.cg.b32 %0, [%1];"
        : "=f"(ret)
        : "l"(ptr)
    );
    return ret;
}

__global__ void L2cacheBW_myDefine_kernel(const float *Arr, float *out) {
    int offset = (BLK_SIZE * UNROLL * blockIdx.x + threadIdx.x) % N_DATA;
    const float *ldg_ptr = Arr + offset;
    float sum = 0;

    #pragma unroll
    for (int i = 0; i < UNROLL; ++i) {
        sum += ldg_cg(ldg_ptr + BLK_SIZE * i);
    }

    if(sum != 0){
        *out = sum;
    }
}

int main(){
    HostPtr<float> arr_h(N_DATA);
    arr_h.SetZeros();
    CuPtr<float> arr_d(arr_h);
    CuPtr<float> y_d(N_DATA);

    std::cout << "Warming Up ..." << std::endl;
    // warm up to cache data into L2
    for (int i = 0; i < WARMUP_ITER; ++i) {
        L2cacheBW_myDefine_kernel<<<GRD_SIZE, BLK_SIZE>>>(arr_d.GetPtr(), y_d.GetPtr());
    }

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float time_ms;
    std::cout << "Running ..." << std::endl;
    checkCudaErrors(hipEventRecord(start));
    for (int i = 0; i < BENCH_ITER ; ++i) {
        L2cacheBW_myDefine_kernel<<<GRD_SIZE, BLK_SIZE>>>(arr_d.GetPtr(), y_d.GetPtr());
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&time_ms, start, stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    double gbps = ((double)((GRD_SIZE*BLK_SIZE*UNROLL) * sizeof(float)) / 1e9) /
                  ((double)time_ms / BENCH_ITER / 1e3);
    printf("L2 cache bandwidth: %lfGB/s\n", gbps);
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L2cacheBW_myDefine L2cacheBW_myDefine.cu

// 驱动调试
// nvcc --keep --keep-dir midRes_myDefine -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L2cacheBW_myDefine L2cacheBW_myDefine.cu
// cuasm --bin2asm midRes_myDefine/L2cacheBW_myDefine.sm_86.cubin -o midRes_myDefine/L2cacheBW_myDefine.sm_86.cuasm
// rm midRes_myDefine/*
// cuasm --asm2bin midRes_myDefine/L2cacheBW_myDefine.sm_86.cuasm -o midRes_myDefine/L2cacheBW_myDefine.sm_86.cubin 


// GRD_SIZE90000的结果
// L2 cache bandwidth: 1276.236447GB/s
// GRD_SIZE9000的结果
// L2 cache bandwidth: 1126.735069GB/s

#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hostptr.hpp"

#include <iostream>

#define GRD_SIZE 900
#define BLK_SIZE 512
#define WARP_SIZE 32
// MAX Bytes for L2_SIZE is 3 * 1024 * 1024 B
#define L2_SIZE (GRD_SIZE * BLK_SIZE)

#define WARMUP_ITER 100
#define BENCH_ITER 100

float posArray[L2_SIZE];

__host__ void assignData(){
    for(int i = 0; i < L2_SIZE; i++){
        posArray[i] = i;
    }
}

// 与访存延迟相比，浮点加消耗的时间可以忽略不计。
__global__ void l2_bw1(float *dsink, float *posArray){
    // 块和线程索引 
    uint32_t tid = threadIdx.x;
    uint32_t bid = blockIdx.x;
    float sink = 0;
    // 从二级缓存中加载数据并累加
    for (uint32_t i = 0; i < L2_SIZE; i += BLK_SIZE) {
        float* ptr = posArray + i;
        // 每次warp都会加载二级缓存中的所有数据
        #pragma unroll
        for (uint32_t j = 0; j < BLK_SIZE; j += WARP_SIZE){
            uint32_t offset = (tid+j)%BLK_SIZE;
            asm volatile ("{\t\n"
                ".reg .f32 data;\n\t" 
                "ld.global.cg.f32 data, [%1];\n\t"
                "add.f32 %0, data, %0;\n\t"
                "}" : "+f"(sink) : "l"(ptr+offset) : "memory"
            );
        }
    }
    if(bid == 0)
        dsink[tid] = sink;
}

// 与访存延迟相比，浮点加消耗的时间可以忽略不计。
__global__ void l2_bw2(float *dsink, float *posArray){
    // 块和线程索引 
    uint32_t tid = threadIdx.x;
    uint32_t bid = blockIdx.x;
    float sink = 0;
    // 从二级缓存中加载数据并累加
    for (uint32_t i = 0; i < L2_SIZE; i += BLK_SIZE) {
        float* ptr = posArray + ((i + bid * BLK_SIZE) % L2_SIZE);
        // 每次warp都会加载二级缓存中的所有数据
        #pragma unroll
        for (uint32_t j = 0; j < BLK_SIZE; j += WARP_SIZE){
            uint32_t offset = (tid+j)%BLK_SIZE;
            asm volatile ("{\t\n"
                ".reg .f32 data;\n\t" 
                "ld.global.cg.f32 data, [%1];\n\t"
                "add.f32 %0, data, %0;\n\t"
                "}" : "+f"(sink) : "l"(ptr+offset) : "memory"
            );
        }
    }
    if(bid == 0)
        dsink[tid] = sink;
}

__global__ void l2_bw3(float *dsink, float *posArray){
    // 块和线程索引 
    uint32_t tid = threadIdx.x;
    uint32_t bid = blockIdx.x;
    float sink = 0;
    // 从二级缓存中加载数据并累加
    for (uint32_t i = 0; i < L2_SIZE; i += BLK_SIZE) {
        float* ptr = posArray + ((i + bid * BLK_SIZE) % L2_SIZE);
        // 每次warp都会加载二级缓存中的所有数据
        float reg[BLK_SIZE/WARP_SIZE];
        #pragma unroll
        for (uint32_t j = 0; j < BLK_SIZE; j += WARP_SIZE){
            uint32_t offset = (tid+j)%BLK_SIZE;
            float ret;
            asm volatile (
                "ld.global.cg.b32 %0, [%1];"
                : "=f"(ret)
                : "l"(ptr+offset)
            );
            reg[j/WARP_SIZE] = ret;
            // asm volatile ("{\t\n"
            //     ".reg .f32 data;\n\t" 
            //     "ld.global.cg.f32 data, [%1];\n\t"
            //     "add.f32 %0, data, %0;\n\t"
            //     "}" : "+f"(sink) : "l"(ptr+offset) : "memory"
            // );
        }
        #pragma unroll
        for (uint32_t j = 0; j < BLK_SIZE/WARP_SIZE; j += 1){
            sink += reg[j];
        }

    }
    if(bid == 0)
        dsink[tid] = sink;
}


int main(){
    assignData();

    CuPtr<float> d_arr(L2_SIZE);
    CuPtr<float> dsink(BLK_SIZE);
    
    unsigned arrBytes = sizeof(float) * L2_SIZE;
    checkCudaErrors(hipMemcpy(d_arr.GetPtr(), posArray, arrBytes, hipMemcpyHostToDevice));

    std::cout << "WarmUp ing ..." << std::endl;
    for(int i = 0; i < WARMUP_ITER; i++){
        l2_bw1<<<GRD_SIZE, BLK_SIZE>>>(dsink.GetPtr(), d_arr.GetPtr());
    }
    hipDeviceSynchronize();

    hipEvent_t startE, stopE;
    float elapsedTime;
    checkCudaErrors(hipEventCreate(&startE));
    checkCudaErrors(hipEventCreate(&stopE));
    std::cout << "Running ..." << std::endl;
    checkCudaErrors(hipEventRecord(startE, 0));
    for(int i = 0; i < BENCH_ITER; i++){
        l2_bw1<<<GRD_SIZE, BLK_SIZE>>>(dsink.GetPtr(), d_arr.GetPtr());
    }
    checkCudaErrors(hipEventRecord(stopE, 0));
    checkCudaErrors(hipEventSynchronize(stopE));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, startE, stopE));
    checkCudaErrors(hipEventDestroy(startE));
    checkCudaErrors(hipEventDestroy(stopE));

    std::cout << "elapsedTime: " << elapsedTime << "ms" << std::endl;
  
    HostPtr<float> hsink;
    dsink.ToHostPtr(hsink);
    float sink_total = 0.0;
    for(int i = 0; i < BLK_SIZE; i++){
        sink_total += hsink(i);
    }
    std::cout << "sink_total: " << sink_total << std::endl;

    // 每个warp都会加载一级缓存中的所有数据
    unsigned long long  BytesAll = L2_SIZE * sizeof(float) * (GRD_SIZE * BLK_SIZE / WARP_SIZE); 
    // 理论上限 16个LD_ST单元 * 30个SM * 4B/LD_ST单元/周期 * 1702M = 3.26784TB/s
    float Bandwidth = (BytesAll / (elapsedTime/BENCH_ITER)) / 1000000;
    
    std::cout << "L2 data cache Bandwidth is: " << Bandwidth << "GB/s" << std::endl;
    
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L2cacheBW_my2 L2cacheBW_my2.cu


// nvcc --keep --keep-dir midRes_paper -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L2cacheBW_my2 L2cacheBW_my2.cu

// cuasm --bin2asm midRes_paper/L2cacheBW_my2.sm_86.cubin midRes_paper/L2cacheBW_my2.sm_86.cuasm


// GRD_SIZE 9000的结果
// elapsedTime: 255733ms
// sink_total: 1.69868e+14
// L2 data cache Bandwidth is: 1037.88GB/s

// GRD_SIZE 900的结果
// elapsedTime: 2263.39ms
// sink_total: 1.69869e+12
// L2 data cache Bandwidth is: 1172.67GB/s
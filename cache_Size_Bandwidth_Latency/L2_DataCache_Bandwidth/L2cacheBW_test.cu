#include "hip/hip_runtime.h"
#include "cuptr.hpp"
#include "hostptr.hpp"
#include "hip/hip_runtime_api.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

// accessed data size in byte, should be smaller than l2 cache size
const size_t DATA_SIZE_IN_BYTE = (1lu << 20) * 2;
// number of LDG instructions 希望它是 （SM数 * 一个SM能启动的BLK_SIZE数） 的倍数
const size_t N_LDG = (1lu << 20) * 90 * 8;
const size_t N_DATA = DATA_SIZE_IN_BYTE / sizeof(float);

#define WARMUP_ITER 200
#define BENCH_ITER 200

#define UNROLL 16
#define BLK_SIZE 512
#define GRD_SIZE (N_LDG / UNROLL / BLK_SIZE)

__device__ __forceinline__
float ldg_cg(const void *ptr) {
    float ret;
    asm volatile (
        "ld.global.cg.f32 %0, [%1];"
        : "=f"(ret)
        : "l"(ptr)
    );
    return ret;
}

__global__ void L2cacheBW_test_kernel(const float *x, float *y) {
    int offset = (BLK_SIZE * UNROLL * blockIdx.x + threadIdx.x) % N_DATA;
    const float *ldg_ptr = x + offset;

    float sum = 0;
    // 每个元素都得用，要不然编译器优化，就不会产生读指令
    #pragma unroll
    for (int i = 0; i < UNROLL; ++i) {
        sum += ldg_cg(ldg_ptr + BLK_SIZE * i);
    }
    // 欺骗编译器？
    if(sum != 0){
        *y = sum;
    }
}

void doTest(){
    static_assert(N_DATA >= UNROLL * BLK_SIZE && N_DATA % (UNROLL * BLK_SIZE) == 0,
                  "UNROLL or BLK_SIZE is invalid");
    
    HostPtr<float> x_h(N_DATA);
    x_h.SetZeros();
    CuPtr<float> x(x_h);
    CuPtr<float> y(N_DATA);

    std::cout << "Warming Up ..." << std::endl;
    // warm up to cache data into L2
    for (int i = 0; i < WARMUP_ITER; ++i) {
        L2cacheBW_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(x.GetPtr(), y.GetPtr());
    }

    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float time_ms;
    std::cout << "Running ..." << std::endl;
    checkCudaErrors(hipEventRecord(start));
    for (int i = 0; i < BENCH_ITER ; ++i) {
        L2cacheBW_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(x.GetPtr(), y.GetPtr());
    }
    checkCudaErrors(hipEventRecord(stop));
    checkCudaErrors(hipEventSynchronize(stop));
    checkCudaErrors(hipEventElapsedTime(&time_ms, start, stop));
    checkCudaErrors(hipEventDestroy(start));
    checkCudaErrors(hipEventDestroy(stop));

    double gbps = ((double)(N_LDG * sizeof(int)) / 1e9) /
                  ((double)time_ms / BENCH_ITER / 1e3);
    printf("L2 cache bandwidth: %fGB/s\n", gbps);
}

int main(){
    doTest();
    return 0;
}


// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L2cacheBW_test L2cacheBW_test.cu

// 驱动调试
// nvcc --keep --keep-dir midRes_test -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/L2cacheBW_test L2cacheBW_test.cu
// cuasm --bin2asm midRes_test/L2cacheBW_test.sm_86.cubin -o midRes_test/L2cacheBW_test.sm_86.cuasm
// rm midRes_test/*
// cuasm --asm2bin midRes_test/L2cacheBW_test.sm_86.cuasm -o midRes_test/L2cacheBW_test.sm_86.cubin 
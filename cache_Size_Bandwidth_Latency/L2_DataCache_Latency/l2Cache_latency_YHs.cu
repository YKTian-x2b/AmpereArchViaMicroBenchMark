
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>

const int WARMUP = 100;
// number of LDG instructions to be timed
const int ROUND = 10;
// stride in byte between LDG instructions
const int STRIDE = 128;

template <int ROUND>
__global__ __launch_bounds__(32, 1)
void l2_latency_kernel(const uint32_t *stride,
                       uint32_t *ret,
                       uint32_t *clk) {
    const char *ldg_ptr = reinterpret_cast<const char *>(stride + threadIdx.x);
    uint32_t val;

    // populate TLB
    asm volatile (
        "ld.global.cg.b32 %0, [%1];\n"
        : "=r"(val)
        : "l"(ldg_ptr)
        : "memory"
    );

    ldg_ptr += val;

    uint32_t start;
    uint32_t stop;

    asm volatile (
        "bar.sync 0;\n"
        "mov.u32 %0, %%clock;\n"
        : "=r"(start) : : "memory"
    );

    #pragma unroll
    for (int i = 0; i < ROUND; ++i) {
        asm volatile (
            "ld.global.cg.b32 %0, [%1];\n"
            : "=r"(val)
            : "l"(ldg_ptr)
            : "memory"
        );

        /*
         * dependent LDG instructions to make sure that
         * LDG latency can not be hidden by parallel LDG.
         *
         * IADD/IMAD/XMAD's latency is much lower than
         * l2 cache and can be ignored.
         */
        ldg_ptr += val;
    }

    asm volatile (
        "bar.sync 0;\n"
        "mov.u32 %0, %%clock;\n"
        : "=r"(stop) : : "memory"
    );

    clk[threadIdx.x] = stop - start;

    // dummy write back
    if (val == 0) {
        *ret = val;
    }
}

int main() {
    static_assert(STRIDE >= 32 * sizeof(uint32_t) &&
                  STRIDE % sizeof(uint32_t) == 0,
                  "invalid 'STRIDE'");

    const uint32_t STRIDE_MEM_SIZE = (ROUND + 1) * STRIDE;

    uint32_t *h_stride;
    hipHostMalloc(&h_stride, STRIDE_MEM_SIZE, hipHostMallocDefault);

    for (int i = 0; i < STRIDE_MEM_SIZE / sizeof(uint32_t); ++i) {
        h_stride[i] = STRIDE;
    }

    uint32_t *d_stride, *d_ret;
    hipMalloc(&d_stride, STRIDE_MEM_SIZE);
    hipMalloc(&d_ret, sizeof(uint32_t));
    hipMemcpy(d_stride, h_stride, STRIDE_MEM_SIZE, hipMemcpyHostToDevice);

    uint32_t *d_clk;
    hipMalloc(&d_clk, 32 * sizeof(uint32_t));

    // pupulate l0/l1 i-cache and l2 cache
    for (int i = 0; i < WARMUP; ++i) {
        l2_latency_kernel<ROUND><<<1, 32>>>(d_stride, d_ret, d_clk);
    }

    // l2 cache latency benchmark
    l2_latency_kernel<ROUND><<<1, 32>>>(d_stride, d_ret, d_clk);

    uint32_t h_clk[32];
    hipMemcpy(h_clk, d_clk, 32 * sizeof(uint32_t), hipMemcpyDeviceToHost);
    printf("l2 cache latency %u cycles\n", h_clk[0] / ROUND);

    hipFree(d_stride);
    hipFree(d_ret);
    hipFree(d_clk);
    hipHostFree(h_stride);

    return 0;
}


// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/l2Cache_latency_YHs l2Cache_latency_YHs.cu

// nvcc --keep --keep-dir YHs_midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/l2Cache_latency_YHs l2Cache_latency_YHs.cu
// cuasm --bin2asm YHs_midRes/l2Cache_latency_YHs.sm_86.cubin YHs_midRes/l2Cache_latency_YHs.sm_86.cuasm
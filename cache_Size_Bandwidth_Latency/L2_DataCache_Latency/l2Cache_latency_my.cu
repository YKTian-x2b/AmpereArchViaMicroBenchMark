#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "hostptr.hpp"
#include "cuptr.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

#define GRD_SIZE 1
#define BLK_SIZE 32
#define WARM_UP 100
#define UNROLL 10
#define ARR_SIZE 1024 * 256
#define STRIDE 16

__device__ __forceinline__
uint32_t ldg_cg(const void *ptr){
    uint32_t ret;
    asm volatile(
        "ld.global.cg.b32 %0, [%1];\n\t"
        : "=r"(ret)
        : "l"(ptr)
        : "memory"
    );
    return ret;
}

__global__ void l2Cache_latency_test_kernel(uint32_t *startClk, uint32_t *stopClk, uint32_t *out, uint32_t* arr){
    int tid = threadIdx.x;
    uint32_t offset = tid;
    uint32_t start, stop;

    // pupulate L2 TLB
    offset = ldg_cg(arr+offset);

    asm volatile (
        "bar.sync 0;\n"
        "mov.u32 %0, %%clock;\n"
        : "=r"(start) : : "memory"
    );

    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        offset = ldg_cg(arr+offset);
    }

    asm volatile (
        "bar.sync 0;\n"
        "mov.u32 %0, %%clock;\n"
        : "=r"(stop) : : "memory"
    );

    startClk[tid] = start;
    stopClk[tid] = stop;

    if(offset == ~0x0){
        *out = offset;
    }
}

int main(){
    HostPtr<uint32_t> arr_h(ARR_SIZE);
    uint32_t *arr_ptr = arr_h.GetPtr();
    for(int i = 0; i < ARR_SIZE; i++){
        arr_ptr[i] = (i + STRIDE) % ARR_SIZE;
    }
    CuPtr<uint32_t> arr_d(arr_h);

    CuPtr<uint32_t> out_d(BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);

    // pupulate l0/l1 i-cache  l2 d-cache
    for(int i = 0; i < WARM_UP; i++){
        l2Cache_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    }
    hipDeviceSynchronize();

    // 
    l2Cache_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();

    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);

    double clockCycles_avg = 0.0;
    for(int i = 0; i < BLK_SIZE; i++){
        // std::cout << startClk_h(i) << " " << stopClk_h(i) << std::endl;
        clockCycles_avg += stopClk_h(i) - startClk_h(i);
    }
    clockCycles_avg /= (BLK_SIZE*UNROLL);

    printf("l2 cache load latency: %lf cycles\n", clockCycles_avg);
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/l2Cache_latency_my l2Cache_latency_my.cu

// nvcc --keep --keep-dir midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/l2Cache_latency_my l2Cache_latency_my.cu
// cuasm --bin2asm midRes/l2Cache_latency_my.sm_86.cubin midRes/l2Cache_latency_my.sm_86.cuasm
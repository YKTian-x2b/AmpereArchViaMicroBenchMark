#include "hip/hip_runtime.h"
#include "cuptr.hpp"
#include "hostptr.hpp"
#include "hip/hip_runtime_api.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>

#define GRD_SIZE 1
#define BLK_SIZE 32
#define UNROLL 10
#define WARM_UP 100

__global__ void l2Cache_latency_test_kernel(uint32_t *startClk, uint32_t *stopClk, void **out, void **arr){
    int tid = threadIdx.x;
    void ** ldg_ptr = arr + tid;

    // pupulate L2 TLB
    asm volatile (
        "ld.global.cg.b64 %0, [%0];\n"
        : "+l"(ldg_ptr)
        : : "memory"
    );


    uint32_t start, stop;
    asm volatile ("bar.sync 0;");
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");

    #pragma unroll
    for(int i = 0; i < UNROLL; i++){
        asm volatile (
            "ld.global.cg.b64 %0, [%0];\n"
            : "+l"(ldg_ptr)
            : : "memory"
        );
    }  
    
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    startClk[tid] = start;
    stopClk[tid] = stop;

    if(ldg_ptr == nullptr){
        *out = ldg_ptr;
    }
}

int main(){
    CuPtr<void*> arr_d(BLK_SIZE);
    HostPtr<void*> arr_h(BLK_SIZE);
    void **h_ptr = arr_h.GetPtr();
    void **d_ptr = arr_d.GetPtr();
    for(int i = 0; i < BLK_SIZE; i++){
        h_ptr[i] = (void*)(d_ptr + i);
    }
    uint32_t arrBytes = BLK_SIZE * sizeof(void*);
    checkCudaErrors(hipMemcpy(d_ptr, h_ptr, arrBytes, hipMemcpyHostToDevice));

    CuPtr<void*> out_d(BLK_SIZE);
    CuPtr<uint32_t> startClk_d(BLK_SIZE);
    CuPtr<uint32_t> stopClk_d(BLK_SIZE);

    // warmup i-cache and l2 cache
    for(int i = 0; i < WARM_UP; i++){
        l2Cache_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    }
    hipDeviceSynchronize();

    // 
    l2Cache_latency_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(startClk_d.GetPtr(), stopClk_d.GetPtr(), out_d.GetPtr(), arr_d.GetPtr());
    hipDeviceSynchronize();

    HostPtr<uint32_t> startClk_h;
    HostPtr<uint32_t> stopClk_h;
    startClk_d.ToHostPtr(startClk_h);
    stopClk_d.ToHostPtr(stopClk_h);

    double clockCycles = stopClk_h(0) - startClk_h(0);
    clockCycles /=  UNROLL;
    std::cout << "l2 cache access latency: " << clockCycles << " cycles" << std::endl;
}

// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/l2Cache_latency_my l2Cache_latency_my.cu

// nvcc --keep --keep-dir midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../../Utils -L /usr/local/cuda/lib64 -l cuda -o res/l2Cache_latency_my l2Cache_latency_my.cu
// cuasm --bin2asm midRes/l2Cache_latency_my.sm_86.cubin midRes/l2Cache_latency_my.sm_86.cuasm

// l2 cache access latency: 221.9 cycles
#include "hip/hip_runtime.h"
#include "../Common.cuh"
#include <iostream>
#include <random>
#include <ctime>
#include "hipblas.h"

// Bytes for total L1/SharedMem 128*1024
#define L1_SIZE 96 * 256
#define THREADS_NUM 512
#define WARP_SIZE 32

float posArray[L1_SIZE];

__host__ void assignData(){
    std::default_random_engine e;
    e.seed(time(0));
    std::uniform_real_distribution<float> u(1, 65536);
    for(int i = 0; i < L1_SIZE; i++){
        posArray[i] = i;
    }
}

__global__ void l1_bw( uint32_t *startClk, uint32_t *stopClk, float *dsink, float *posArray){
    // 线程索引
    uint32_t tid = threadIdx.x;
    // 侧效变量，目的是避免编译器删除这段代码
    float sink = 0;
    // 通过填充 L1 缓存来预热
    for (uint32_t i = tid; i < L1_SIZE; i += THREADS_NUM) {
        float * ptr = posArray+i;
        // sink += *ptr;
        asm volatile ("{\t\n"
        ".reg .f32 data;\n\t"
        "ld.global.ca.f32 data, [%1];\n\t"
        "add.f32 %0, data, %0;\n\t"
        "}" : "+f"(sink) : "l"(ptr): "memory"
        );
    }
    // 同步所有线程
    asm volatile ("bar.sync 0;");
    // 开始计时 uint32_t
    uint32_t start = 0;
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start) :: "memory");
    // 从 L1 缓存加载数据，累加
    for (uint32_t i = 0; i < L1_SIZE; i += THREADS_NUM) {
        float * ptr = posArray+i;
        // 每个warp都会加载一级缓存中的所有数据
        for (uint32_t j = 0; j < THREADS_NUM; j += WARP_SIZE) {
            uint32_t offset = (tid+j)%THREADS_NUM;
            // sink += ptr[offset];
            asm volatile ("{\t\n"
            ".reg .f32 data;\n\t" 
            "ld.global.ca.f32 data, [%1];\n\t"
            "add.f32 %0, data, %0;\n\t"
            "}" : "+f"(sink) : "l"(ptr+offset) : "memory"
            );
        }
    }
    // 同步所有线程
    asm volatile ("bar.sync 0;");
    // 停止计时 uint32_t
    uint32_t stop = 0;
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");
    // 将时间和数据写回内存
    startClk[tid] = start;
    stopClk[tid]= stop;
    dsink[tid]= sink;
}

int main(){
    assignData();

    unsigned arrBytes = sizeof(float) * L1_SIZE;
    unsigned clkBytes = sizeof(uint32_t) * THREADS_NUM;
    unsigned sinkBytes = sizeof(float) * THREADS_NUM;

    uint32_t *d_startClk, *d_stopClk;
    float *d_arr, *dsink;
    uint32_t *h_startClk, *h_stopClk;
    float *hsink;

    CHECK(hipMalloc(&d_arr, arrBytes));
    CHECK(hipMalloc(&d_startClk, clkBytes));
    CHECK(hipMalloc(&d_stopClk, clkBytes));
    CHECK(hipMalloc(&dsink, sinkBytes));
    CHECK(hipMemcpy(d_arr, posArray, arrBytes, hipMemcpyHostToDevice));

    l1_bw<<<1, THREADS_NUM>>>(d_startClk, d_stopClk, dsink, d_arr);
    hipDeviceSynchronize();
   
    h_startClk = (uint32_t*)malloc(clkBytes);
    h_stopClk = (uint32_t*)malloc(clkBytes);
    hsink = (float*)malloc(sinkBytes);
    CHECK(hipMemcpy(h_startClk, d_startClk, clkBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_stopClk, d_stopClk, clkBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(hsink, dsink, sinkBytes, hipMemcpyDeviceToHost));

    float clockCycles_avg = 0.0;
    float sink_total = 0.0;
    for(int i = 0; i < THREADS_NUM; i++){
        clockCycles_avg += h_stopClk[i] - h_startClk[i];
        // std::cout << h_startClk[i] << " " << h_stopClk[i] << std::endl;
        sink_total += hsink[i];
    }
    clockCycles_avg /= THREADS_NUM;
    std::cout << h_startClk[0] << " " << h_stopClk[0] << std::endl;
    std::cout << "clockCycles_avg: " << clockCycles_avg << std::endl; // 63433
    std::cout << "sink_total: " << sink_total << std::endl;
    // 每个warp都会加载一级缓存中的所有数据 一个SM启动了 128/32个warp 所以一共读取了BytesAll个字节 per SM
    uint32_t BytesAll = L1_SIZE * sizeof(float) * (THREADS_NUM / WARP_SIZE); 
    // 理论上限是64B 实测结果是62B左右
    float BpCpSM = BytesAll / clockCycles_avg;
    std::cout << "L1 real throughput is  " << BpCpSM <<  " Bytes per cycle per SM: " << std::endl;

    CHECK(hipFree(d_arr));
    CHECK(hipFree(d_startClk));
    CHECK(hipFree(d_stopClk));
    CHECK(hipFree(dsink));
    free(h_startClk);
    free(h_stopClk);
    free(hsink);
}

// nvcc L1cacheMB_bkp.cu -o res/L1cacheMB_bkp -gencode=arch=compute_86,code=\"sm_86,compute_86\"
// 
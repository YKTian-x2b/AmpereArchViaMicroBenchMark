
#include <hip/hip_runtime.h>
// #include "../Common.cuh"
// #include <iostream>
// #include <random>
// #include <ctime>
// #include "cublas_v2.h"

// #define BLOCK_XY 64

// #define STRIDE 64

// #define ARRSIZE 2048
// int arr[ARRSIZE];

// __host__ void assignData(){
//     std::default_random_engine e;
//     e.seed(time(0));
//     std::uniform_real_distribution<int> u(-65536, 65536);
//     for(int i = 0; i < ARRSIZE; i++){
//         arr[i] = u(e);
//     }
// }

// // for L1 cache
// __global__ void L1cacheMB(int * arr){
//     int idx = blockIdx.x * blockDim.x + threadIdx.x;
//     int x = arr[idx];
// }

// float L1cacheMB_run(int * d_arr, cudaEvent_t &event_start, cudaEvent_t &event_stop)
// {
//     dim3 blockDim(BLOCK_XY);
//     dim3 gridDim(1);

//     float elapsedTime;
//     CHECK(cudaEventRecord(event_start, 0));

//     L1cacheMB<<<gridDim, blockDim>>>(d_arr);

//     CHECK(cudaEventRecord(event_stop, 0));
//     CHECK(cudaEventSynchronize(event_stop));

//     CHECK(cudaEventElapsedTime(&elapsedTime, event_start, event_stop));
//     return elapsedTime;
// }

// int main(){
//     assignData();
//     int * d_arr;
//     unsigned arrBytes = sizeof(int) * ARRSIZE;
//     CHECK(cudaMalloc(&d_arr, arrBytes));
//     CHECK(cudaMemcpy(d_arr, A, arrBytes, cudaMemcpyHostToDevice));

//     cudaEvent_t event_start, event_stop;
//     CHECK(cudaEventCreate(&event_start));
//     CHECK(cudaEventCreate(&event_stop));

//     float elapsedTime = regbank_test_run();

//     CHECK(cudaEventDestroy(event_start));
//     CHECK(cudaEventDestroy(event_stop));
    
    
// }
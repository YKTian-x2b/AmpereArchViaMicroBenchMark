#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>
using namespace std;

typedef unsigned int uint;

#define GRD_SIZE (1800)
#define BLK_SIZE (128)
#define WARP_CUT (4)
#define N_ITER (256)
#define N_UNROLL (256)
#define N_WARMUP (3)
#define N_TEST (5)

__global__ void regbank_test_kernel(const int2 c, const int NIter, const float4 v, float* a)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int wid = tid / 32;
    int lid = tid % 32;

    float v0 = v.x;
    float v1 = v.y;
    float v2 = v.z;


    for(int i=0; i<NIter; i++)
    {
        #pragma unroll
        for(int n=0; n<N_UNROLL; n++)
            v0 = fmaf(v0, v2, v1);
    }
    

    __syncthreads();
    
    // only first lane of warp in first block writes to memory
    if(bid ==0 && lid==0)
        a[wid] = v0;
}

float regbank_test_run(const int2 c, const int NIter, const float4 v, float* a, hipEvent_t &event_start, hipEvent_t &event_stop)
{
    float elapsedTime;
    checkCudaErrors(hipEventRecord(event_start, 0));

    regbank_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(c, NIter, v, a);

    checkCudaErrors(hipEventRecord(event_stop, 0));
    checkCudaErrors(hipEventSynchronize(event_stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, event_start, event_stop));

    return elapsedTime;
}

float regbank_test_run_drv(const int2 c, const int NIter, const float4 v, float* a, hipEvent_t &event_start, hipEvent_t &event_stop)
{
    static hipModule_t cuModule;
    static hipFunction_t kernel;
    static bool isInitialized = false;

    if (!isInitialized)
    {
        hipInit(0);

        // Create module from binary file
        hipModuleLoad(&cuModule, "midRes/regBankTest.sm_86.cubin");

        // Get function handle from module _Z19regbank_test_kernel4int2i6float4Pf
        hipModuleGetFunction(&kernel, cuModule, "_Z19regbank_test_kernel4int2i6float4Pf");

        // printf("cuModule = %#llx\n", (unsigned long long)cuModule);
        // printf("cuFunction = %#llx\n", (unsigned long long)kernel);
        isInitialized = true;
    }

    float elapsedTime;
    checkCudaErrors(hipEventRecord(event_start, 0));

    void* args[] = { (void*)&c, (void*)&NIter, (void*)&v, (void*)&a };
    hipModuleLaunchKernel(kernel,
                   GRD_SIZE, 1, 1, BLK_SIZE, 1, 1,
                    0, 0, args, 0);
    //regbank_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(c, NIter, v, a);

    checkCudaErrors(hipEventRecord(event_stop, 0));
    checkCudaErrors(hipEventSynchronize(event_stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, event_start, event_stop));

    return elapsedTime;
}

void dotest()
{
    CuPtr<float> da(4096);
    int NIter = N_ITER;

    hipEvent_t event_start, event_stop;
    checkCudaErrors(hipEventCreate(&event_start));
    checkCudaErrors(hipEventCreate(&event_stop));

    //double giga_instr = (1e-9 * GRD_SIZE) * BLK_SIZE * N_ITER * N_UNROLL;

    int2 c = make_int2(WARP_CUT, 0);
    float4 v = make_float4(0, 1.0f, 0, 0);

    printf("### Warm up...\n");
    for(int i=0; i<N_WARMUP; i++)
    {
        da.SetZeros();
        float elapsedTime = regbank_test_run_drv(c, NIter, v, da.GetPtr(), event_start, event_stop); // in ms
        // float elapsedTime = regbank_test_run(c, NIter, v, da.GetPtr(), event_start, event_stop);
        printf("  Warmup %2d: %10.3f ms\n", i, elapsedTime);
    }
    
    printf("### Testing...\n");
    for(int i=0; i<N_TEST; i++)
    {
        da.SetZeros();
        float elapsedTime = regbank_test_run_drv(c, NIter, v, da.GetPtr(), event_start, event_stop); // in ms
        // float elapsedTime = regbank_test_run(c, NIter, v, da.GetPtr(), event_start, event_stop);
        printf("  Test %2d: %10.3f ms\n", i, elapsedTime);
    }

    checkCudaErrors(hipEventDestroy(event_start));
    checkCudaErrors(hipEventDestroy(event_stop));

    printf("\n### Result checking...\n");

    HostPtr<float> ha;
    da.ToHostPtr(ha);
    for(int i=0; i<BLK_SIZE/32; i++)
    {
        unsigned int xa = *(unsigned int *)(&ha(i));
        printf("res[%2d] : %8g  0x%08x\n", i, ha(i), xa);
    }
}

int main()
{
    dotest();
    return 0;
}


// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I/opt/kaiProjects/GEMM_kai/Utils -L /usr/local/cuda/lib64 -l cuda -o res/regBankTest regBankTest.cu



// nvcc --keep --keep-dir midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I/opt/kaiProjects/GEMM_kai/Utils -L /usr/local/cuda/lib64 -l cuda -o res/regBankTest regBankTest.cu

// cuasm --bin2asm midRes/regBankTest.sm_86.cubin -o midRes/regBankTest.sm_86.cuasm

// cp midRes/regBankTest.sm_86.cuasm res/regBankTest.template.sm_86.cuasm && cp midRes/regBankTest.sm_86.cuasm res/regBankTest.origin.sm_86.cuasm

// @CUASM_INSERT_MARKER_POS.

// python3 test_regBank.py
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>
using namespace std;

typedef unsigned int uint;

#define GRD_SIZE 1800
#define BLK_SIZE 512
#define UNROLL 10
#define N_WARMUP 10
#define N_TEST 5

__global__ void regbank_test_kernel(const float4 v, float *out)
{
    int tid = threadIdx.x;
    float v0 = v.x;
    float v1 = v.y;
    float v2 = v.z;

    #pragma unroll
    for(int n = 0; n < UNROLL; n++){
        v0 = fmaf(v0, v2, v1);
    }

    if(v0 == 0){
        out[tid] = v0;
    }  
}

float regbank_test_run(const float4 v, float *out, hipEvent_t &event_start, hipEvent_t &event_stop)
{
    float elapsedTime;
    checkCudaErrors(hipEventRecord(event_start, 0));

    regbank_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(v, out);

    checkCudaErrors(hipEventRecord(event_stop, 0));
    checkCudaErrors(hipEventSynchronize(event_stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, event_start, event_stop));

    return elapsedTime;
}

float regbank_test_run_drv(const float4 v, float *out, hipEvent_t &event_start, hipEvent_t &event_stop)
{
    static hipModule_t cuModule;
    static hipFunction_t kernel;
    static bool isInitialized = false;

    if (!isInitialized)
    {
        hipInit(0);

        // Create module from binary file
        hipModuleLoad(&cuModule, "midRes/regBankTest_2Bank.sm_86.cubin");

        // Get function h_outndle from module 
        hipModuleGetFunction(&kernel, cuModule, "_Z19regbank_test_kernel6float4Pf");

        isInitialized = true;
    }

    float elapsedTime;
    checkCudaErrors(hipEventRecord(event_start, 0));

    void* args[] = {(void*)&v, (void*)&out};
    hipModuleLaunchKernel(kernel,
                   GRD_SIZE, 1, 1, BLK_SIZE, 1, 1,
                    0, 0, args, 0);

    checkCudaErrors(hipEventRecord(event_stop, 0));
    checkCudaErrors(hipEventSynchronize(event_stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, event_start, event_stop));

    return elapsedTime;
}

void dotest()
{
    CuPtr<float> d_out(BLK_SIZE);

    hipEvent_t event_start, event_stop;
    checkCudaErrors(hipEventCreate(&event_start));
    checkCudaErrors(hipEventCreate(&event_stop));

    float4 v = make_float4(0, 1.0f, 0, 0);

    printf("### Warm up...\n");
    for(int i=0; i<N_WARMUP; i++)
    {
        d_out.SetZeros();
        float elapsedTime = regbank_test_run_drv(v, d_out.GetPtr(), event_start, event_stop); // in ms
        // float elapsedTime = regbank_test_run(v, d_out.GetPtr(), event_start, event_stop);
        // printf("  Warmup %2d: %10.3f ms\n", i, elapsedTime);
    }
    
    printf("### Testing...\n");
    float elapsed_avg = 0.0;
    for(int i=0; i<N_TEST; i++)
    {
        d_out.SetZeros();
        float elapsedTime = regbank_test_run_drv(v, d_out.GetPtr(), event_start, event_stop); // in ms
        // float elapsedTime = regbank_test_run(v, d_out.GetPtr(), event_start, event_stop);
        // printf("  Test %2d: %10.3f ms\n", i, elapsedTime);
        elapsed_avg += elapsedTime;
    }
    printf("elapsedTime_avg: %10.3fms\n", elapsed_avg/N_TEST);

    checkCudaErrors(hipEventDestroy(event_start));
    checkCudaErrors(hipEventDestroy(event_stop));

    printf("\n### Result checking...\n");

    HostPtr<float> h_out;
    d_out.ToHostPtr(h_out);
    for(int i=0; i<BLK_SIZE/32; i++)
    {
        unsigned int xa = *(unsigned int *)(&h_out(i));
        // printf("res[%2d] : %8g  0x%08x\n", i, h_out(i), xa);
    }
}

int main()
{
    dotest();
    return 0;
}


// nvcc -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I/opt/kaiProjects/GEMM_kai/Utils -L /usr/local/cuda/lib64 -l cuda -o res/regBankTest_2Bank regBankTest_2Bank.cu




// nvcc --keep --keep-dir midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I/opt/kaiProjects/GEMM_kai/Utils -L /usr/local/cuda/lib64 -l cuda -o res/regBankTest_2Bank regBankTest_2Bank.cu

// cuasm --bin2asm midRes/regBankTest_2Bank.sm_86.cubin -o midRes/regBankTest_2Bank.sm_86.cuasm

// cp midRes/regBankTest_2Bank.sm_86.cuasm res/regBankTest_2Bank.template.sm_86.cuasm && cp midRes/regBankTest_2Bank.sm_86.cuasm res/regBankTest_2Bank.origin.sm_86.cuasm

// @CUASM_INSERT_MARKER_POS.

// python3 test_regBank_2Bank.py
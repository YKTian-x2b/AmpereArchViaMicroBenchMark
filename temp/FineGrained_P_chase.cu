#include "hip/hip_runtime.h"


#define ARR_SIZE 1024
float A[ARR_SIZE];

__host__ void assignData(int stride){
    for(int i = 0; i < ARR_SIZE; i++){
        A[i] = (i+stride) % ARR_SIZE;
    }
}

__global__ void FG_P_chase(int n_iter, float* res){

    __shared__ unsigned int s_tvalue[];
    __shared__ unsigned int s_index[];

    float sum = 0;
    float j = 0;
    for(int it = 0; it < n_iter; it++){
        uint32_t start_time;
        asm volatile ("mov.u32 %0, %%clock;" :"=r"(start_time) :: "memory");

        j = A[j];
        s_index[it] = j;
        sum += j;

        uint32_t stop_time;
        asm volatile ("mov.u32 %0, %%clock;" :"=r"(stop_time) :: "memory");
        s_tvalue[it] = stop_time - start_time;
    }

    if(){
        *res = sum;
    }
}

int main(){

    std::cout << "WarmUping ..." << std::endl;
    P_chase<<<>>>();
    hipDeviceSynchronize();

    std::cout << "Running ..." << std::endl;
    P_chase<<<>>>();
}
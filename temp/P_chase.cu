#include "hip/hip_runtime.h"


#define ARR_SIZE 1024
float A[ARR_SIZE];

__host__ void assignData(int stride){
    for(int i = 0; i < ARR_SIZE; i++){
        A[i] = (i+stride) % ARR_SIZE;
    }
}

__global__ void P_chase(int n_iter, float* res){

    uint32_t start_time;
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(start_time) :: "memory");

    float sum = 0;
    float j = 0;
    for(int it = 0; it < n_iter; it++){
        j = A[j];
        sum += j;
    }

    uint32_t stop_time;
    asm volatile ("mov.u32 %0, %%clock;" :"=r"(stop_time) :: "memory");

    if(){
        *res = sum;
    }
}

int main(){

    std::cout << "WarmUping ..." << std::endl;
    P_chase<<<>>>();
    hipDeviceSynchronize();

    std::cout << "Running ..." << std::endl;
    P_chase<<<>>>();
}
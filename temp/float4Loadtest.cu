#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hostptr.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstdio>

#define GRD_SIZE 1
#define BLK_SIZE 128
#define N_DATA 1024

// __device__ __forceinline__
// float ldg_cg(const void *ptr) {
//     float ret;
//     asm volatile (
//         "ld.global.cg.f32 %0, [%1];"
//         : "=f"(ret)
//         : "l"(ptr)
//     );
//     return ret;
// }

__global__ void float4Loadtest(float* gloArr, float* res){
    int tid = threadIdx.x;
    float4 *ldg_ptr = (float4 *)(gloArr + tid);
    float4 reg = *ldg_ptr;
    *res = reg.x + reg.y + reg.z + reg.w;

    __syncthreads();
    reg = *(ldg_ptr+1);
    *res = reg.x + reg.y + reg.z + reg.w;
}

int main(){
    CuPtr<float> gloArr(N_DATA+4);
    gloArr.SetZeros();
    CuPtr<float> res(N_DATA);
    float4Loadtest<<<GRD_SIZE, BLK_SIZE>>>(gloArr.GetPtr(), res.GetPtr());
}

// nvcc --keep --keep-dir midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda float4Loadtest.cu -o float4Loadtest
// cuasm --bin2asm midRes/float4Loadtest.sm_86.cubin
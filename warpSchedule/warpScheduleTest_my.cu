#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>
using namespace std;

#define GRD_SIZE 1
// [512, 512+32, 512+64, 512+96, 512+128]
#define BLK_SIZE (512+64)
#define N_UNROLL 10

__global__ void warpSchedule_test_kernel(const float4 v, unsigned *startClk, unsigned *stopClk, float *out)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int bdx = blockDim.x;
    int wid = tid >> 5;
    int lid = tid & 31;

    float v0 = v.x;
    float v1 = v.y;
    float v2 = v.z;
    
    // 开始计时 start
    unsigned start = 0;
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");

    #pragma unroll
    for(int n = 0; n < N_UNROLL; n++){
        v0 = fmaf(v0, v2, v1);
    }
    
    // 停止计时 stop
    unsigned stop = 0;
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    // 将时间和数据写回内存  
    // only first lane of warp writes to memory
    if(lid == 0){
        int index = ((bid * bdx) >> 5) + wid;
        out[index] = v0;
        startClk[index] = start;
        stopClk[index]= stop;
    }  
}

float warpSchedule_test_run(const float4 v, unsigned *startClk, unsigned *stopClk, float *out)
{
    hipEvent_t event_start, event_stop;
    checkCudaErrors(hipEventCreate(&event_start));
    checkCudaErrors(hipEventCreate(&event_stop));
    float elapsedTime;
    checkCudaErrors(hipEventRecord(event_start, 0));

    warpSchedule_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(v, startClk, stopClk, out);

    checkCudaErrors(hipEventRecord(event_stop, 0));
    checkCudaErrors(hipEventSynchronize(event_stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, event_start, event_stop));
    checkCudaErrors(hipEventDestroy(event_start));
    checkCudaErrors(hipEventDestroy(event_stop));
    return elapsedTime;
}

float warpSchedule_test_run_drv(const float4 v, unsigned *startClk, unsigned *stopClk, float *out)
{
    static hipModule_t cuModule;
    static hipFunction_t kernel;
    static bool isInitialized = false;

    if (!isInitialized)
    {
        hipInit(0);

        // Create module from binary file
        hipModuleLoad(&cuModule, "midRes2/warpScheduleTest_my.sm_86.cubin");

        // Get function handle from module
        hipModuleGetFunction(&kernel, cuModule, "_Z24warpSchedule_test_kernel6float4PjS0_Pf");

        isInitialized = true;
    }

    hipEvent_t event_start, event_stop;
    checkCudaErrors(hipEventCreate(&event_start));
    checkCudaErrors(hipEventCreate(&event_stop));
    float elapsedTime;
    checkCudaErrors(hipEventRecord(event_start, 0));

    void* args[] = { (void*)&v, (void*)&startClk, (void*)&stopClk, (void*)&out};
    hipModuleLaunchKernel(kernel,
                   GRD_SIZE, 1, 1, BLK_SIZE, 1, 1,
                    0, 0, args, 0);

    checkCudaErrors(hipEventRecord(event_stop, 0));
    checkCudaErrors(hipEventSynchronize(event_stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, event_start, event_stop));
    checkCudaErrors(hipEventDestroy(event_start));
    checkCudaErrors(hipEventDestroy(event_stop));
    return elapsedTime;
}

void doTest()
{
    size_t eleSize = GRD_SIZE * BLK_SIZE / 32;
    CuPtr<float> d_out(eleSize);
    CuPtr<unsigned> d_startClk(eleSize);
    CuPtr<unsigned> d_stopClk(eleSize);

    float4 v = make_float4(1.0f, 1.0f, 1.0f, 1.0f);

    printf("### Warming Up...\n");
    d_out.SetZeros();
    d_startClk.SetZeros();
    d_stopClk.SetZeros();

    warpSchedule_test_run_drv(v, d_startClk.GetPtr(), d_stopClk.GetPtr(), d_out.GetPtr());
    // warpSchedule_test_run(v, d_startClk.GetPtr(), d_stopClk.GetPtr(), d_out.GetPtr());
    
    printf("### Runing...\n");
    d_out.SetZeros();
    d_startClk.SetZeros();
    d_stopClk.SetZeros();

    float elapsedAll = warpSchedule_test_run_drv(v, d_startClk.GetPtr(), d_stopClk.GetPtr(), d_out.GetPtr());
    // float elapsedAll = warpSchedule_test_run(v, d_startClk.GetPtr(), d_stopClk.GetPtr(), d_out.GetPtr());
    
    printf("\n### Result checking...\n");
    HostPtr<float> h_out;
    HostPtr<unsigned> h_startClk;
    HostPtr<unsigned> h_stopClk;
    d_out.ToHostPtr(h_out);
    d_startClk.ToHostPtr(h_startClk);
    d_stopClk.ToHostPtr(h_stopClk);
    for(int i = 0; i < eleSize; i++){
        unsigned elapsed = h_stopClk(i)-h_startClk(i);
        printf("index[%2d],  res: %8.3f, warpIdx: %d,  startClk: %10u,  elapsed %10uclks\n", i, h_out(i), i%4, h_startClk(i), elapsed);
    }
    printf("elapsed_all: %10fms\n", elapsedAll);
}

int main(){
    doTest();
    return 0;
}

// nvcc --keep --keep-dir midRes2 -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/warpScheduleTest_my warpScheduleTest_my.cu

// cuasm --bin2asm midRes2/warpScheduleTest_my.sm_86.cubin -o midRes2/warpScheduleTest_my.sm_86.cuasm

// cp midRes2/warpScheduleTest_my.sm_86.cuasm res/warpScheduleTest_my.template.sm_86.cuasm && cp midRes2/warpScheduleTest_my.sm_86.cuasm res/warpScheduleTest_my.origin.sm_86.cuasm

// @CUASM_INSERT_MARKER_POS.

// python3 test_warpSchedule_my.py
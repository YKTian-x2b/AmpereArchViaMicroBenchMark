#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>
using namespace std;

#define GRD_SIZE (1)
#define BLK_SIZE (256)
#define N_UNROLL (10)

__global__ void L0CacheSize_test_kernel(const float4 v, unsigned *startClk, unsigned *stopClk, float *a, int warpAIdx, int warpBIdx)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int bdx = blockDim.x;
    int wid = tid >> 5;
    int lid = tid & 31;

    float v0 = v.x;
    float v1 = v.y;
    float v2 = v.z;
    float v3 = v.w;
    
    // 开始计时 start
    unsigned start = 0;
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");

    if(wid == warpAIdx){
        #pragma unroll
        for(int n = 0; n < N_UNROLL; n++)
            v0 = fmaf(v0, v2, v1);
    }
    else if(wid == warpBIdx){
        #pragma unroll
        for(int n = 0; n < N_UNROLL; n++)
            v0 = fmaf(v0, v2, v3);
    }
    else{
        ;
    }
    
    // 停止计时 stop
    unsigned stop = 0;
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    // 将时间和数据写回内存  
    // only first lane of warp writes to memory
    if(lid == 0){
        int index = (bid * bdx + tid) >> 5;
        a[index] = v0;
        startClk[index] = start;
        stopClk[index]= stop;
    }  
}

float L0CacheSize_test_run(const float4 v, unsigned *startClk, unsigned *stopClk, float *a, int warpAIdx, int warpBIdx)
{
    hipEvent_t event_start, event_stop;
    checkCudaErrors(hipEventCreate(&event_start));
    checkCudaErrors(hipEventCreate(&event_stop));
    float elapsedTime;
    checkCudaErrors(hipEventRecord(event_start, 0));

    L0CacheSize_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(v, startClk, stopClk, a, warpAIdx, warpBIdx);

    checkCudaErrors(hipEventRecord(event_stop, 0));
    checkCudaErrors(hipEventSynchronize(event_stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, event_start, event_stop));
    checkCudaErrors(hipEventDestroy(event_start));
    checkCudaErrors(hipEventDestroy(event_stop));
    return elapsedTime;
}

float L0CacheSize_test_run_drv(const float4 v, unsigned *startClk, unsigned *stopClk, float *a, int warpAIdx, int warpBIdx)
{
    static hipModule_t cuModule;
    static hipFunction_t kernel;
    static bool isInitialized = false;

    if (!isInitialized)
    {
        hipInit(0);

        // Create module from binary file
        hipModuleLoad(&cuModule, "midRes/warpScheduleTest_t4.sm_86.cubin");

        // Get function handle from module
        hipModuleGetFunction(&kernel, cuModule, "_Z23L0CacheSize_test_kernel6float4PjS0_Pfii");

        isInitialized = true;
    }

    hipEvent_t event_start, event_stop;
    checkCudaErrors(hipEventCreate(&event_start));
    checkCudaErrors(hipEventCreate(&event_stop));
    float elapsedTime;
    checkCudaErrors(hipEventRecord(event_start, 0));

    void* args[] = { (void*)&v, (void*)&startClk, (void*)&stopClk, (void*)&a, (void*)&warpAIdx, (void*)&warpBIdx};
    hipModuleLaunchKernel(kernel,
                   GRD_SIZE, 1, 1, BLK_SIZE, 1, 1,
                    0, 0, args, 0);

    checkCudaErrors(hipEventRecord(event_stop, 0));
    checkCudaErrors(hipEventSynchronize(event_stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, event_start, event_stop));
    checkCudaErrors(hipEventDestroy(event_start));
    checkCudaErrors(hipEventDestroy(event_stop));
    return elapsedTime;
}

void doTest()
{
    size_t eleSize = GRD_SIZE * BLK_SIZE / 32;
    CuPtr<float> da(eleSize);
    CuPtr<unsigned> d_startClk(eleSize);
    CuPtr<unsigned> d_stopClk(eleSize);

    float4 v = make_float4(1.0f, 1.0f, 1.0f, 1.0f);

    // printf("### Warming Up...\n");
    da.SetZeros();
    d_startClk.SetZeros();
    d_stopClk.SetZeros();

    for(int warpAIdx = 0; warpAIdx < 4; warpAIdx++){
        for(int warpBIdx = 4; warpBIdx < 8; warpBIdx++){
            L0CacheSize_test_run_drv(v, d_startClk.GetPtr(), d_stopClk.GetPtr(), da.GetPtr(), warpAIdx, warpBIdx);
            // L0CacheSize_test_run(v, d_startClk.GetPtr(), d_stopClk.GetPtr(), da.GetPtr(), warpAIdx, warpBIdx);
        }
    }

    printf("### Runing...\n");
    for(int warpAIdx = 0; warpAIdx < 4; warpAIdx++){
        for(int warpBIdx = 4; warpBIdx < 8; warpBIdx++){
            da.SetZeros();
            d_startClk.SetZeros();
            d_stopClk.SetZeros();

            float elapsedAll = L0CacheSize_test_run_drv(v, d_startClk.GetPtr(), d_stopClk.GetPtr(), da.GetPtr(), warpAIdx, warpBIdx);
            // float elapsedAll = L0CacheSize_test_run(v, d_startClk.GetPtr(), d_stopClk.GetPtr(), da.GetPtr(), warpAIdx, warpBIdx);
            
            // printf("\n### Result checking...\n");
            HostPtr<float> ha;
            da.ToHostPtr(ha);
            HostPtr<unsigned> h_startClk;
            d_startClk.ToHostPtr(h_startClk);
            HostPtr<unsigned> h_stopClk;
            d_stopClk.ToHostPtr(h_stopClk);
            for(int i = 0; i < eleSize; i++)
            {
                // if(i == warpAIdx || i == warpBIdx){
                unsigned elapsed = h_stopClk(i)-h_startClk(i);
                printf("index[%2d],  res: %8.3f,  warpIdx: %d,  startClk: %10u,  elapsed %10uclks\n", i, ha(i), i%4, h_startClk(i), elapsed);
                // }
            }
            printf("elapsed_all: %10fms\n", elapsedAll);
        }
    }    
}

int main(){
    doTest();
    return 0;
}

// nvcc --keep --keep-dir midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I../Utils -L /usr/local/cuda/lib64 -l cuda -o res/warpScheduleTest_t4 warpScheduleTest_t4.cu

// cuasm --bin2asm midRes/warpScheduleTest_t4.sm_86.cubin -o midRes/warpScheduleTest_t4.sm_86.cuasm

// cp midRes/warpScheduleTest_t4.sm_86.cuasm res/warpScheduleTest_t4.template.sm_86.cuasm && cp midRes/warpScheduleTest_t4.sm_86.cuasm res/warpScheduleTest_t4.origin.sm_86.cuasm

// @CUASM_INSERT_MARKER_POS.

// python3 test_warpSchedule_t4.py
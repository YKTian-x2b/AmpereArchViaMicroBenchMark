#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include "cuptr.hpp"
#include "hip/hip_runtime.h"

#include <cstdio>
#include <iostream>
using namespace std;

#define GRD_SIZE (1)
#define BLK_SIZE (256)
#define N_UNROLL (4)

__global__ void L0CacheSize_test_kernel(const float4 v, int n_unroll, unsigned *startClk, unsigned *stopClk, float *a, int warpAIdx, int warpBIdx)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int bdx = blockDim.x;
    int wid = tid >> 5;
    int lid = tid & 31;

    float v0 = v.x;
    float v1 = v.y;
    float v2 = v.z;
    float v3 = v.w;
    
    // 开始计时 start
    unsigned start = 0;
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(start) :: "memory");

    if(wid == warpAIdx){
        #pragma unroll
        for(int n = 0; n < n_unroll; n++)
            v0 = fmaf(v0, v2, v1);
    }
    else if(wid == warpBIdx){
        #pragma unroll
        for(int n = 0; n < n_unroll; n++)
            v0 = fmaf(v0, v2, v3);
    }
    else{
        ;
    }
    
    // 停止计时 stop
    unsigned stop = 0;
    asm volatile ("mov.u32 %0, %%clock;" : "=r"(stop) :: "memory");

    // 将时间和数据写回内存  
    // only first lane of warp writes to memory
    if(lid == 0){
        int index = ((bid * bdx) >> 5) + wid;
        a[index] = v0;
        startClk[index] = start;
        stopClk[index]= stop;
    }  
}

float L0CacheSize_test_run(const float4 v, int n_unroll, unsigned *startClk, unsigned *stopClk, float *a, int warpAIdx, int warpBIdx)
{
    hipEvent_t event_start, event_stop;
    checkCudaErrors(hipEventCreate(&event_start));
    checkCudaErrors(hipEventCreate(&event_stop));
    float elapsedTime;
    checkCudaErrors(hipEventRecord(event_start, 0));

    L0CacheSize_test_kernel<<<GRD_SIZE, BLK_SIZE>>>(v, n_unroll, startClk, stopClk, a, warpAIdx, warpBIdx);

    checkCudaErrors(hipEventRecord(event_stop, 0));
    checkCudaErrors(hipEventSynchronize(event_stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, event_start, event_stop));
    checkCudaErrors(hipEventDestroy(event_start));
    checkCudaErrors(hipEventDestroy(event_stop));
    return elapsedTime;
}

float L0CacheSize_test_run_drv(const float4 v, int n_unroll, unsigned *startClk, unsigned *stopClk, float *a, int warpAIdx, int warpBIdx)
{
    static hipModule_t cuModule;
    static hipFunction_t kernel;
    static bool isInitialized = false;

    if (!isInitialized)
    {
        hipInit(0);

        // Create module from binary file
        hipModuleLoad(&cuModule, "warpScheduleTest.sm_86.cubin");

        // Get function handle from module
        hipModuleGetFunction(&kernel, cuModule, "_Z23L0CacheSize_test_kernel6float4iPjS0_Pfii");

        printf("cuModule = %#llx\n", (unsigned long long)cuModule);
        printf("cuFunction = %#llx\n", (unsigned long long)kernel);
        isInitialized = true;
    }

    hipEvent_t event_start, event_stop;
    checkCudaErrors(hipEventCreate(&event_start));
    checkCudaErrors(hipEventCreate(&event_stop));
    float elapsedTime;
    checkCudaErrors(hipEventRecord(event_start, 0));

    void* args[] = { (void*)&v, (void*)&n_unroll, (void*)&startClk, (void*)&stopClk, (void*)&a, (void*)&warpAIdx, (void*)&warpBIdx};
    hipModuleLaunchKernel(kernel,
                   GRD_SIZE, 1, 1, BLK_SIZE, 1, 1,
                    0, 0, args, 0);

    checkCudaErrors(hipEventRecord(event_stop, 0));
    checkCudaErrors(hipEventSynchronize(event_stop));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, event_start, event_stop));
    checkCudaErrors(hipEventDestroy(event_start));
    checkCudaErrors(hipEventDestroy(event_stop));
    return elapsedTime;
}

void doTest()
{
    size_t eleSize = GRD_SIZE * BLK_SIZE / 32;
    CuPtr<float> da(eleSize);
    CuPtr<unsigned> d_startClk(eleSize);
    CuPtr<unsigned> d_stopClk(eleSize);

    float4 v = make_float4(1.0f, 1.0f, 1.0f, 1.0f);

    printf("### Warming Up...\n");
    da.SetZeros();
    d_startClk.SetZeros();
    d_stopClk.SetZeros();
    int n_unroll = N_UNROLL;

    int warpAIdx = 0;
    int warpBIdx = 4;
    L0CacheSize_test_run_drv(v, n_unroll, d_startClk.GetPtr(), d_stopClk.GetPtr(), da.GetPtr(), warpAIdx, warpBIdx);
    
    printf("### Runing...\n");

    // for(int warpAIdx = 3; warpAIdx < 4; warpAIdx++){
    //     for(int warpBIdx = 4; warpBIdx < 8; warpBIdx++){
            da.SetZeros();
            d_startClk.SetZeros();
            d_stopClk.SetZeros();
        
            float elapsedAll = L0CacheSize_test_run_drv(v, n_unroll, d_startClk.GetPtr(), d_stopClk.GetPtr(), da.GetPtr(), warpAIdx, warpBIdx);
            // L0CacheSize_test_run(v, n_unroll, d_startClk.GetPtr(), d_stopClk.GetPtr(), da.GetPtr(), warpAIdx, warpBIdx);
            
            printf("\n### Result checking...\n");
            HostPtr<float> ha;
            da.ToHostPtr(ha);
            HostPtr<unsigned> h_startClk;
            d_startClk.ToHostPtr(h_startClk);
            HostPtr<unsigned> h_stopClk;
            d_stopClk.ToHostPtr(h_stopClk);
            for(int i = 0; i < eleSize; i++)
            {
                // if(i == warpAIdx || i == warpBIdx){
                    unsigned elapsed = h_stopClk(i)-h_startClk(i);
                    printf("index[%2d],  res: %8.3f, warpIdx: %d,  startClk: %10u,  elapsed %10uclks\n", i, ha(i), i%4, h_startClk(i), elapsed);
                // }
            }
            printf("elapsed_all: %10fms\n", elapsedAll);
    //     }
    // }    
}

int main(){
    doTest();
    return 0;
}

// nvcc --keep -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I/opt/kaiProjects/GEMM_kai/Utils -L /usr/local/cuda/lib64 -l cuda -o warpScheduleTest save/warpScheduleTest.cu

// cuasm --bin2asm warpScheduleTest.sm_86.cubin

// cp warpScheduleTest.sm_86.cuasm save/warpScheduleTest.template.sm_86.cuasm && cp warpScheduleTest.sm_86.cuasm save/warpScheduleTest.origin.sm_86.cuasm

// @CUASM_INSERT_MARKER_POS.WORK_1

// python3 save/test_warpSchedule.py